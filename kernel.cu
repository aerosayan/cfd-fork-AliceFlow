#include "hip/hip_runtime.h"
// AliceFlow_v0_24.cpp 
// 9 ���� 2017 ������� �� 64 ������ ����� int64_t.
// 15 ������ 2017 ��������������� � visual studio community edition 2017 (open source).
// 1 ������� 2016 �������������� �� nvidia cuda 8.0. 
// ���� ����� ������� � ����� ��� ������������� � ������ ����. 
// 11 ������ 2016 ���� ������� cl_agl_amg_v0_14.
// 15 ������� 2015 ����. ������ � Visual Studio 2015.
// AliceFlow_v0_21.cpp
// 15 ������� 2015. ������ � Visual Studio 2013.
// AliceFlow_v0_20.cpp
// 14 ������� 2015 ������������� ���������� ����������������� lusol � ilu2 decomposition �� 2 ������.
// AliceFlow_v0_07.cpp: ���������� ����� ����� ��� ����������� ����������.
// AliceFlow_v0_07.cpp �� ������  AliceFlow_v0_06.cpp, �� ������ � LES ������� ��������������.
// ��������� �� ������ ������������ � �������� ������.
// 17 ������ 2013 ����. ���������� ����������������� lusol_.
// 1 ������ 2013. ������ � Visual Studio 2012.
//
// AliceFlow_v0_06.cpp :
// 3D ��������� AliceFlow_v0_06.cpp ��������� �������� AliceFlowv0_05.cpp
// �������� �� ������, ��������� ������������ � ����������������.
// 
// ��������� AliceFlowv0_05.cpp, 
// ���������� ������� AliceFlowv0_03.cpp, ����������� 
// ���� �� ������� ����������� ��������������
// � ������ ����������� �� ���� �������.
// � ���� �� ��������� ������������ 
// ������������� ����������������� HEX 
// ��������� �����.
// begin one 17 ��� 2011 ����.
//
// 3D ��������� AliceFlowv0_05.cpp 
// ������������ ��������� ��������:
// 1. ��������� ��������������� �����.
// 2. �������� � ���������� ���� ��������
//    ������ ����������� ��� ������ ������ ����.
// 3. ������ ������� ����.
// 4. �������� ����.
// 5. ������� � ������������ tecplot360.
// begin two 30 ���� 2011 ����.
// begin three 14 ������� 2011 ����. ������ �� Visual Studio 2010.
// begin four 12 ����� 2012 ����. (�� ������� ����� �. ������ - ������� �� ���).
//
// �������� ������� ��������� ����� � ����������� ���� TGF2023_*
// ����� 0.2x120��� �������� 100 �������� (10-20��).
// ��� ������ �������� ��� ����� ������� ������� ������� 2-3 ����� ���������, � ����� 3-5 ���� ���������. 

// ���������������� � ������ ���� ������ ���������� �������������� ������������ gcc �� GNU.
//#define MINGW_COMPILLER 1

#ifdef MINGW_COMPILLER
#include <stdio.h>
#endif

// ��� std::locale::global(std::locale("en_US.UTF-8"));
// �� ��������.
//#include <locale.h>

//#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cinttypes> // ��� ���� 64 ������� ������ ����� int64_t


#include <stdlib.h> 
#include <omp.h> // OpenMP
//-Xcompiler "/openmp" 

//using namespace System;

// ������������ ����������.
#define doubleprecision 1
#if doubleprecision == 1
#define doublereal double //double // ������ ������������� ����� ������� ��������
//#define doublereal Decimal // decimal
#else
#define doublereal float //float // ������ ������������� ����� ��������� ��������
#endif

#define doubleintprecision 1



#if doubleintprecision == 1
#define integer int64_t
#else
#define integer int
#endif

// 9 september 2017.
// ������ �� ������������ ����������� ������ � ����� ���������� �������� ������.
// �������� �������� �������� ��� ������������ �����, �.�. ����� ������ ��� �������� �� ������, � ���� ���������� 
// ������� ������. ������ ������� ������ �������� ������� �� ������������� ���� ���������� ��� ����� ����� � 
// ����������� ������������������ �������� ���������� �� ����� �����.
integer ireconstruction_free_construct_alloc = 1; // 0 - off, 1 - on.
// ���������� �� ��������� � ��������� ����
// �� ��������� ������� ������ ���� �� �������.
integer ianimation_write_on = 0; // 0 - off, 1 - on.

// ��� ���������� ���������� �� ������ 300 �������� ������� �� ��������� vel*rGradual_changes � ����� 
// ������ ������������������� � ����������� ��� �� ��������� vel.
doublereal rGradual_changes = 0.1; // 1.0 - �� ������������.

// ������������� ��������� �������� ����������.
// ������ �������� ��� ���� ��������� �������.
// initialization value.
doublereal starting_speed_Vx = 0.0;
doublereal starting_speed_Vy = 0.0;
doublereal starting_speed_Vz = 0.0;

// ������� ����� ��� XY-Plot (variation Plot).
// �� ��������� ������� �����, ����� ������� �������� �����, � ���������� 
// ����� ����� ����� �� ���� ���������� ������������� ������� ���������.
doublereal Tochka_position_X0_for_XY_Plot = 0.0;
doublereal Tochka_position_Y0_for_XY_Plot = 0.0;
doublereal Tochka_position_Z0_for_XY_Plot = 0.0;
integer idirectional_for_XY_Plot = 0; // 0 - Ox axis. 

// ��� iVar==TEMP && lw==1 ����� �� ������� ����� ������������ ����� ������������ ����������� ����� V ������� ���������� ����� 0.5K.
bool bPhysics_stop = false;
// ������ ��������� ���������� ����������� ������ ��� ����.
bool bPhysics_PTBSH_memory = false;
// ������ ������ ������������� � ������ ���� :
bool bonly_solid_calculation = false;

// 3 ������� 2015 ����� ����� �������� ����� GUI ������������
// � ����� � ��� ���������� �������������� �������� � ����� ������ ����.
// ������������ �����
#define UNEVEN_MUSCL 1017  // van Leer (1977)
#define UNEVEN_SOUCUP 1018 // MINMOD
#define UNEVEN_HLPA 1019
#define UNEVEN_SMART 1020 // Gaskell and Lau (1988)
#define UNEVEN_WACEB 1021
#define UNEVEN_SMARTER 1022
#define UNEVEN_STOIC 1023 // Darwish (1993)
#define UNEVEN_CLAM 1024
#define UNEVEN_OSHER 1025 // Chakravarthy and Osher (1983)
#define UNEVEN_VONOS 1026
#define UNEVEN_LPPA 1027
#define UNEVEN_EXPONENTIAL 1028
#define UNEVEN_SUPER_C 1029
#define UNEVEN_ISNAS 1030
#define UNEVEN_CUBISTA 1031
#define UNEVEN_GAMMA 1032 // ����� � ���������� beta_m
#define UNEVEN_COPLA 1033 // 1 08 2015
#define UNEVEN_SECBC 1034 // 2 08 2015 Yu et al., (2001b) ��������, �������.
#define UNEVEN_SGSD 1035 // 3 08 2015 Li and Tao (2002)

// ���������� �������������� ������������� ������� �� ����������.
typedef struct TMY_AMG_MANAGER {

	// 0 - �� �������� ������� �������, 
	// 1 - �������� ������� �������.
	integer bTemperatureMatrixPortrait;
	integer bSpeedMatrixPortrait;
	integer bPressureMatrixPortrait;
	integer bStressMatrixPortrait;
	integer bMatrixPortrait;

	// fgmres(m_restart)
	integer m_restart;

	// lfil for BiCGStab+ILU2 and fgmres.
	integer lfil;

	// Temperature
	doublereal theta_Temperature;
	integer maximum_delete_levels_Temperature;
	integer nFinnest_Temperature, nu1_Temperature, nu2_Temperature;
	integer memory_size_Temperature;
	integer ilu2_smoother_Temperature; // 0 - �� ������������, 1 - ������������.
	// 0 - AVL Tree, 1 - SPLAY Tree, 2 - Binary Heap, 3 - Treap.
	// default - 3.
	integer iCFalgorithm_and_data_structure_Temperature;
	// Speed
	doublereal theta_Speed;
	integer maximum_delete_levels_Speed;
	integer nFinnest_Speed, nu1_Speed, nu2_Speed;
	integer memory_size_Speed;
	integer ilu2_smoother_Speed; // 0 - �� ������������, 1 - ������������.
	// 0 - AVL Tree, 1 - SPLAY Tree, 2 - Binary Heap, 3 - Treap.
	// default - 3.
	integer iCFalgorithm_and_data_structure_Speed;
	// Pressure
	doublereal theta_Pressure;
	integer maximum_delete_levels_Pressure;
	integer nFinnest_Pressure, nu1_Pressure, nu2_Pressure;
	integer memory_size_Pressure;
	integer ilu2_smoother_Pressure; // 0 - �� ������������, 1 - ������������.
	// 0 - AVL Tree, 1 - SPLAY Tree, 2 - Binary Heap, 3 - Treap.
	// default - 3.
	integer iCFalgorithm_and_data_structure_Pressure;
	// Stress
	doublereal theta_Stress;
	integer maximum_delete_levels_Stress;
	integer nFinnest_Stress, nu1_Stress, nu2_Stress;
	integer memory_size_Stress;
	integer ilu2_smoother_Stress; // 0 - �� ������������, 1 - ������������.
	// 0 - AVL Tree, 1 - SPLAY Tree, 2 - Binary Heap, 3 - Treap.
	// default - 3.
	integer iCFalgorithm_and_data_structure_Stress;
	// global
	bool bCFJacoby;
	integer iRunge_Kutta_smoother; // 3 - �������� �������, 5 - ������ �������, ����� ������ ����� �� ������������. 
	integer iFinnest_ilu; // 0 �� ������������, 1 - ilu0. ������ �� ����� ��������� �����.
	// ������������� iluk ���������� �� �������� ������� ����������� ��� �������
	// �������� ������ nnz/n ����� ������ ������ ���� ������ 6 (�����).
	bool b_ilu_smoothers_in_nnz_n_LE_6;
	doublereal theta; // strength threshold
	//integer maximum_levels; // ������������ ���������� ������� ����������� (������ ���� ������������).
	integer maximum_delete_levels; // ���������� ������� ���������� ����� � ������� ������ �����.
	integer nFinnest, nu1, nu2; // ���������� �����������.
	// 0 - AVL Tree, 1 - SPLAY Tree, 2 - Binary Heap, 3 - Treap.
	// default - 3.
	integer iCFalgorithm_and_data_structure;
	integer memory_size; // � �������� ������� �.
	// ��� ������ ������� ���������� � ������������.
	integer number_interpolation_procedure; // ������������� ��������� �������������.
	integer number_interpolation_procedure_Temperature;
	integer number_interpolation_procedure_Speed;
	integer number_interpolation_procedure_Pressure;
	integer number_interpolation_procedure_Stress;

	// 6 december 2016.
	integer itypemodifyinterpol; // ����� ����������� �������������.
	integer inumberadaptpass; // ������������ ���������� ������-�������� � �������������.
	doublereal gold_const, gold_const_Temperature, gold_const_Speed, gold_const_Pressure, gold_const_Stress;
	doublereal magic;
	doublereal F_to_F_Temperature, F_to_F_Speed, F_to_F_Pressure, F_to_F_Stress; // magic
	integer ilu2_smoother; // 0 - �� ������������, 1 - ������������.
	// AMG Splitting (coarsening)
	// ������ ���������� C-F ��������� : 0 - standart, 1 - RS 2.
	// RS 2 ���������� ������ ���������� C-F ��������� ���������� ������ ������.
	integer icoarseningTemp, icoarseningSpeed, icoarseningPressure, icoarseningStress;
	integer icoarseningtype;
	// Stabilization BiCGStab.
	// 8.01.2017
	integer istabilizationTemp, istabilizationSpeed, istabilizationPressure, istabilizationStress;
	integer istabilization;
	// ipatch - ����� �����.
	integer ipatch_number;

	integer iprint_log, iprint_log_Temperature, iprint_log_Speed, iprint_log_Pressure, iprint_log_Stress;

	// truncation for interpolation.
	integer itruncation_interpolation, itruncation_interpolation_Temperature, itruncation_interpolation_Speed, itruncation_interpolation_Pressure, itruncation_interpolation_Stress;
	double truncation_interpolation, truncation_interpolation_Temperature, truncation_interpolation_Speed, truncation_interpolation_Pressure, truncation_interpolation_Stress;

	// gmres smoother
	// �.����, ����� [1986].
	bool b_gmresTemp, b_gmresSpeed, b_gmresPressure, b_gmresStress;
	bool b_gmres;

} MY_AMG_MANAGER;

MY_AMG_MANAGER my_amg_manager;

bool bglobal_first_start_radiation = true;

// ���� �� ������ �������������� ������ ������������� � ������� ����.
bool bglobal_unsteady_temperature_determinant = false;

// ����� ��������� ���������� :
// simplemeshgen == 0 ��� unevensimplemeshgen ==1.
// �� ��������� ������������ simplemeshgen == 0.
integer iswitchMeshGenerator = 0; // ������� �������� ���������.
// �������������� ��� ������������ �������������.
integer steady_or_unsteady_global_determinant = 0; // 0 - steady, 1- unsteady.

// ������������ �� ���������� �������� ������������ ��������� �����.
bool b_on_adaptive_local_refinement_mesh = false;
integer itype_ALICE_Mesh = 1;// ��� ���� �����.

typedef struct TTimeStepLaw
{
	integer id_law; // 0 - Linear, 1 - Square Wave.
	doublereal Factor_a_for_Linear;
	doublereal tau; // ������������ �������� ��� Square Wave
	// 06_03_2017 ���������� ����� ���� � �������.
	doublereal Q; // ���������� ��� Square Wave.
	// ���������� ����� ��� ���� �������.
	doublereal m1, tau1, tau2, tau_pause, T_all;
	integer n_cycle;
	// hot cold reshime (double linear)
	doublereal on_time_double_linear;

} TimeStepLaw;

TimeStepLaw glTSL;

// 24 ������� 2016. 
// ��� ��������� ����� ���������� ����� � ����� 0.14 ��������.
typedef struct TQuickNonlinearBoundaryCondition {
	doublereal emissivity;
	doublereal Tamb, dS;
	doublereal film_coefficient;
	bool bactive;
	bool bStefanBolcman_q_on;
	bool bNewtonRichman_q_on;

} QuickNonlinearBoundaryCondition;

QuickNonlinearBoundaryCondition* qnbc = NULL;
integer iadd_qnbc_maxelm = 0; // ��� ����������� ������
bool b_sign_on_nonlinear_bc = false;


// ������� �� �� SIMPLE ����������.
// ��� ����� ��� ����� ������ ��������� ������� ��� ��������� �������������.
// ������ BiCGStab_internal3 ��������.
bool bSIMPLErun_now_for_temperature = false;
// ��� ����� ��� ����� ������ ��������� ������� ��� ��������� �������������
// ��� ������� amg1r5 ���������� ����� � ����������� ����������.
bool bSIMPLErun_now_for_natural_convection = false;
// �������������� ������ ���������� ��� �����������.
doublereal* told_temperature_global_for_HOrelax = NULL;

/*
��� ���������� ������� ���������� ����� ������������ ����������� �����.
��� ����� ����� ��� ���� �� ��������������� ��-����� �� ���������.
������� ������������� ������ ��������� � ���� ��� ���������������� ����� ������ ���� �����������
��� ��������� ����� ����������� ������� ����������� � ������ �����. � �������� ����������������
������ ������� ��������������.
*/
bool *sourse2Dproblem=NULL;
doublereal *conductivity2Dinsource=NULL;

// �������������� ������ ����������.
bool bHORF = false;
bool bdontstartsolver = false;
doublereal* bPamendment_source_old = NULL;
doublereal* bsource_term_radiation_for_relax = NULL;
doublereal* b_buffer_correct_source = NULL;
// �� ��������� ������������ �����_0.14 �� ������ �� ��������.
bool bfirst_start_nonlinear_process = true;

// ������� �������-������� �� ������� ��� �����������.
integer adiabatic_vs_heat_transfer_coeff = 0; // 0 - adiabatic wall, 1 - Newton Richman condition, 2 - Stefan Bolcman condition, 3 - mix condition.
// ��� ���������� ��������� ������� �� ������ ���� ������ ���� ��������� ����� V ������. 
bool breakRUMBAcalc_for_nonlinear_boundary_condition = false;
bool bvacuumPrism = false; // ������� ��������� �����������.
bool bdouble_vacuum_PRISM = false; // ������� ��������� ����������. ��� ����� ��� ��������� ���������� ������� - ��������� �� ������ ������.
bool bBlockStefanBolcman = false; // ���� true �� ��������� ������� ���������.
doublereal film_coefficient = 0.0; // ����������� �����������.
doublereal operating_temperature_for_film_coeff = 20.0; // Tamb for Newton-Richman condition.
// ���� ��������� ���� ���������� ����� ���������������� ��������� operating_temperature_for_film_coeff
// �� � ������� �������-������� �� ������� ������� �������� ����� (���������� ������� �������) ��� ���
// ������� �������� �������������� � ���������� ������� ������� ������� � ����������� ������������ � ������������
// ��������������� ��������. ����� ����� �������� ������������ ���������� blocker_Newton_Richman.
bool blocker_Newton_Richman = true;

FILE* fp_radiation_log = NULL;
errno_t err_radiation_log;

// 1 - ������������ ������ ������� ����.
// 0 - ������������ � �������� � �������� ����.
integer ionly_solid_visible = 0;

// ������������ ����� �������������� ������������� ������� � ���������� ��� ��� ������ BiCGStab+ILU2.
// 0 - �������� BiCGStab + ILU2. 1 - �������� ��������������� �������������� ������ amg1r5.
// 2 - BiCGStab + ADI (Lr1sk).
integer iswitchsolveramg_vs_BiCGstab_plus_ILU2 = 0; // BiCGStab + ILU2.

bool bwait = false; // ���� false �� �� ���������� getchar().
#define admission 1.0e-10 // ��� ����������� ���������� ���� ������������ �����.

unsigned int calculation_vorst_seach_time = 0;

// ���� ����������� ������ �������� 
// ����������� � 200 �������� ������� 
// �� ������ ������ �� ����� (������).
// � ������ ���������� ����������� ������ TEMPERATURE_FAILURE_DC
// �� ������� ���������� ��������������� ��������� � ����� ����
// ��� ������������ ������ ��������� �������������� ����� �� ���������.
#define TEMPERATURE_FAILURE_DC 5000.2


// �������� ����� ��������� ��������� ������� �������� �������������
// ���������� �������. ��. ��� ����������� � ����� ������ �. ���������.
// BETA 1.0 4/3=1.333333333 6/5=1.2
#define BETA 1.0

// UDS  ��. my_approx_convective.c
unsigned int iFLOWScheme = 2; // ��������������� ������� �������
unsigned int iTEMPScheme = 2; // ��������������� ������� �������

// �������� ����� ������ ���������� �������� SIMPLEC
// SIMPLEC Van Doormal and Raithby, 1984 ���.
// SIMPLEC ���������� �� SIMPLE ������ � ���� �����:
// 1. � SIMPLEC �� ������������ ������ ���������� ��� �������� ��� ��������� ��������, �.�. alphaP=1.0.
// 2. � SIMPLEC ������ ����� ��������������� tau ~ alpha/(1-alpha), � � SIMPLE tau ~ alpha. 
// � ��������� ��������� ��������� ���������.
#define SIMPLE_Carretto 0 // �������� SIMPLE Carretto et al., 1973 ������������ �� ���������.
#define SIMPLEC_Van_Doormal_and_Raithby 1 // �������� SIMPLEC Van Doormal and Raithby, 1984 ���.
unsigned int iSIMPLE_alg = SIMPLE_Carretto;// SIMPLE_Carretto SIMPLEC_Van_Doormal_and_Raithby

// �������� ������� ���� ��� ���� ������� ��� �����
// (residual,residual) ��� () ��������� ������������ � R^n.
// �������� ���� ��������� ������ �������� �� �������� ������������� 
// ����. ��������� � ������ ����������� �� ��������� ����� �� �������
// ������ ������������ ������ (FVM). ������� ��� ��������� ����� ��������������
// ������������� ������� � ������ mysolverv0_03.c. ��� ������������� ���������� ��,
// ��� ���� �� ����� ������ ������ ������ ��� ������������ �������� ����������� �������,
// ����� ������� ��� ����������� �������� �������� ������� ��� ������� ������ �������� �������������
// �� ������ ����� �������� ��������� (����������). ��� ����� ���������.
// ��� ���������� ������ �������� ������� �������������������� ������� 1.0e-4
// �������� ���������� ������ �� CFX �� �������.
// ���� ����� �������� �� ������� ���������� ����� 1.0e-3.
// �������� ����� �� ������ �� CFX.
doublereal dterminatedTResudual = 1e-5; // ��� ��� Congruate Gradients � ����� BiCGStab_internal3.

doublereal globalEndTimeUnsteadyTemperatureCalculation = 1.0; // ���������� ����� ��� �������������� ������������� ������������� � ������ ����. 

// � ���� ���� ����� ������������ ���������� � ������
// �������� ��������� ����.
FILE *fp_statistic_convergence=NULL;
// � ���� ���� ����� �������������� ������ ����.
// ��� ����� ��� ������� ��������� �������� � ���������.
// �� �������� ���������� ��� ���� �������� ���������� � ���������� ��������������� ��������.
FILE *fp_log=NULL;

// ������������ ��� ��������� ������� 
// ������ ������������ �������������.
bool bsolid_static_only = false;

const integer inumcore = 2; // ����� ���� ����������
const bool bparallelizm_old = false;

// ��������� ������ ������� :
typedef struct TPARBOUND {
	integer ileft_start, ileft_finish, iright_start, iright_finish, iseparate_start, iseparate_finish;
	bool active; // ���������� ������������.
} PARBOUND;


// ��������� ������ ������������ ��� �����������������.
typedef struct TPARDATA {
	integer ncore; // 1, 2, 4, 8.
	integer *inumerate=NULL;
	// ��� ��� ncore==2;
	PARBOUND b0;
	// ��� ��� ncore==4;
	PARBOUND b00, b01;
	// ��� ��� ncore==8;
	PARBOUND b000, b001, b010, b011;
} PARDATA;

PARDATA nd;




// ������������ � ����� ��� ��������� �����.
doublereal* rthdsd_no_radiosity_patch = NULL;



#include "adaptive_local_refinement_mesh.cpp" // ����
#include "constr_struct.cpp" // ���������� �������� ������ TEMPER � FLOW

// ���������� ������, ���������� � ������.
integer lb, ls, lw;
BLOCK* b = NULL; // ������
SOURCE* s = NULL; // ����������
WALL* w = NULL; // ������ ������

doublereal *xpos = NULL, *ypos = NULL, *zpos = NULL;
doublereal *xposadd = NULL, *yposadd = NULL, *zposadd = NULL;

#include "my_LR.c" // ������������ �����

#include "my_material_properties.c" // ���������� �������� ������� ����������

// ������������� ����������� ��������� ���������-��������
// �� ����������� �����
#include "pamendment3.c"


#include "shortest_distance.cpp" // ���������� ����������� ���������� �� ������

// 8 ������ 2016.
const bool bvery_big_memory = true; // true ��� ������ � ���� �� ������ � ����������� ������. ��� ����������� ������� �� ��������.

struct Tdatabase {
	doublereal *x=NULL, *y=NULL, *z=NULL; // ���������� �����.
	integer maxelm;
	integer** nvtxcell=NULL;
	integer ncell;
	// ����� ������������� � ��������������.
	integer **ptr=NULL;// ��� ������������ ��������������� �������������� ������
};

Tdatabase database;

TEMPER t;
integer flow_interior; // ��������� ����� FLUID ���
FLOW* f=NULL;

// ������� �������� � ��������� tecplot360
#include "my_export_tecplot3.c"

// ���������� � ������� ���������� ������� ������������ �� 
// icepak user guide.
typedef struct TFLUENT_RESIDUAL{
	// ������ ������� ���������� �� ������ ����� ������� ����.
	// ������� ������������� � ���������� FLUENT
	// �.�. ����������� �� ������� fluent.
	doublereal res_vx; // ������� X ��������
	doublereal res_vy; // ������� Y ��������
	doublereal res_vz; // ������� Z ��������
	doublereal res_no_balance; // ������������������ ��������� �����.
	doublereal operating_value_b; // �������� ������������������ ���������� ����� � ���������� ��������.
} FLUENT_RESIDUAL;


// ���������� ��������
#include "mysolverv0_03.c"




// �������������� ������ ��� �����������
// �� ������ ������������� �������,
// � ����� �������������� ������ ��� 
// ������������� �� ������ ������������� �������.
#include "my_unsteady_temperature.c"

// ������������� ��� ������������ ���������.
#include "my_nested_dissection.cpp"

#include <ctime> // ��� ������ ������� ����������.




integer ltdp; // ���������� �������� �������� ��������� �� ����������� � �������� �����.
TEMP_DEP_POWER* gtdps=NULL; // Garber temperature depend power sequence. 

// ���� ������ ����������:
integer lmatmax; // ������������ ����� ����������
TPROP* matlist=NULL; // ��������� ���� ������ ����������





doublereal rterminate_residual_ICCG_Oh2(FLOW floc) {
	// �������� ������������� O(h!2)
	doublereal* resterm = new doublereal[floc.maxelm + floc.maxbound];
	for (integer i = 0; i<floc.maxelm + floc.maxbound; i++) {
		resterm[i] = 0.0; // �������������.
	}

	for (integer iP = 0; iP<floc.maxelm; iP++) {
		// ���������� �������� �������� ������������ ������:
		doublereal dx = 0.0, dy = 0.0, dz = 0.0;// ����� �������� ������������� ������
		volume3D(iP, floc.nvtx, floc.pa, dx, dy, dz);
		doublereal dl = fmin(dx, fmin(dy, dz));
		resterm[iP] = 0.1*dl*dl; // O(h!2)
		integer iE, iN, iT, iW, iS, iB; // ������ �������� ����������� �������
		iE = floc.sosedi[ESIDE][iP].iNODE1; iN = floc.sosedi[NSIDE][iP].iNODE1; iT = floc.sosedi[TSIDE][iP].iNODE1; iW = floc.sosedi[WSIDE][iP].iNODE1; iS = floc.sosedi[SSIDE][iP].iNODE1; iB = floc.sosedi[BSIDE][iP].iNODE1;
		// ���� � ����� �� ������ ����� ������� ��������� �������
		// �� ��������������� ���������� ����� true
		bool bE = false, bN = false, bT = false, bW = false, bS = false, bB = false;

		if (iE >= floc.maxelm) bE = true;
		if (iN >= floc.maxelm) bN = true;
		if (iT >= floc.maxelm) bT = true;
		if (iW >= floc.maxelm) bW = true;
		if (iS >= floc.maxelm) bS = true;
		if (iB >= floc.maxelm) bB = true;

		if ((bE) || (bW)) {
			dl = 0.5*dx;
			if (bE) resterm[iE] = 0.1*dl*dl; // O(h!2)
			if (bW) resterm[iW] = 0.1*dl*dl; // O(h!2)
		}
		if ((bN) || (bS)) {
			dl = 0.5*dy;
			if (bN) resterm[iN] = 0.1*dl*dl; // O(h!2)
			if (bS) resterm[iS] = 0.1*dl*dl; // O(h!2)
		}
		if ((bT) || (bB)) {
			dl = 0.5*dz;
			if (bT) resterm[iT] = 0.1*dl*dl; // O(h!2)
			if (bB) resterm[iB] = 0.1*dl*dl; // O(h!2)
		}
	}
	doublereal ret = Scal(resterm, resterm, floc.maxelm + floc.maxbound);
	delete[] resterm;
	resterm=NULL;
	return ret;
} // rterminate_residual_ICCG_Oh2

doublereal rterminate_residual_LR1sk_Oh3(FLOW floc) {
	// �������� ������������� O(h!2)
	doublereal* resterm = new doublereal[floc.maxelm + floc.maxbound];
	for (integer i = 0; i<floc.maxelm + floc.maxbound; i++) {
		resterm[i] = 0.0; // �������������.
	}

	for (integer iP = 0; iP<floc.maxelm; iP++) {
		// ���������� �������� �������� ������������ ������:
		doublereal dx = 0.0, dy = 0.0, dz = 0.0;// ����� �������� ������������� ������
		volume3D(iP, floc.nvtx, floc.pa, dx, dy, dz);
		doublereal dl = fmin(dx, fmin(dy, dz));
		resterm[iP] = 0.1*dl*dl*dl; // O(h!3)
		integer iE, iN, iT, iW, iS, iB; // ������ �������� ����������� �������
		iE = floc.sosedi[ESIDE][iP].iNODE1; iN = floc.sosedi[NSIDE][iP].iNODE1; iT = floc.sosedi[TSIDE][iP].iNODE1; iW = floc.sosedi[WSIDE][iP].iNODE1; iS = floc.sosedi[SSIDE][iP].iNODE1; iB = floc.sosedi[BSIDE][iP].iNODE1;
		// ���� � ����� �� ������ ����� ������� ��������� �������
		// �� ��������������� ���������� ����� true
		bool bE = false, bN = false, bT = false, bW = false, bS = false, bB = false;

		if (iE >= floc.maxelm) bE = true;
		if (iN >= floc.maxelm) bN = true;
		if (iT >= floc.maxelm) bT = true;
		if (iW >= floc.maxelm) bW = true;
		if (iS >= floc.maxelm) bS = true;
		if (iB >= floc.maxelm) bB = true;

		if ((bE) || (bW)) {
			dl = 0.5*dx;
			if (bE) resterm[iE] = 0.1*dl*dl*dl; // O(h!3)
			if (bW) resterm[iW] = 0.1*dl*dl*dl; // O(h!3)
		}
		if ((bN) || (bS)) {
			dl = 0.5*dy;
			if (bN) resterm[iN] = 0.1*dl*dl*dl; // O(h!3)
			if (bS) resterm[iS] = 0.1*dl*dl*dl; // O(h!3)
		}
		if ((bT) || (bB)) {
			dl = 0.5*dz;
			if (bT) resterm[iT] = 0.1*dl*dl*dl; // O(h!3)
			if (bB) resterm[iB] = 0.1*dl*dl*dl; // O(h!3)
		}
	}
	doublereal ret;
	//ret=Scal(resterm,resterm,floc.maxelm+floc.maxbound);
	ret = NormaV(resterm, floc.maxelm + floc.maxbound);
	// ������������ ����������� ������.
	if (resterm != NULL) {
		delete[] resterm;
		resterm = NULL;
	}
	return ret;
} // rterminate_residual_LR1sk_Oh3

doublereal rterminate_residual_LR1sk_temp_Oh3(TEMPER t) {
	// �������� ������������� O(h!2)
	// �������� ������������� ������������� � ������ ���� O(h).
	doublereal* resterm = new doublereal[t.maxelm + t.maxbound];
	for (integer i = 0; i<t.maxelm + t.maxbound; i++) {
		resterm[i] = 0.0; // �������������.
	}

	for (integer iP = 0; iP<t.maxelm; iP++) {
		// ���������� �������� �������� ������������ ������:
		doublereal dx = 0.0, dy = 0.0, dz = 0.0;// ����� �������� ������������� ������
		volume3D(iP, t.nvtx, t.pa, dx, dy, dz);
		doublereal dl = fmin(dx, fmin(dy, dz));
		//resterm[iP]=0.1*dl*dl*dl; // O(h!3)
		resterm[iP] = dl; // O(h)
		integer iE, iN, iT, iW, iS, iB; // ������ �������� ����������� �������
		iE = t.sosedi[ESIDE][iP].iNODE1; iN = t.sosedi[NSIDE][iP].iNODE1; iT = t.sosedi[TSIDE][iP].iNODE1; iW = t.sosedi[WSIDE][iP].iNODE1; iS = t.sosedi[SSIDE][iP].iNODE1; iB = t.sosedi[BSIDE][iP].iNODE1;
		// ���� � ����� �� ������ ����� ������� ��������� �������
		// �� ��������������� ���������� ����� true
		bool bE = false, bN = false, bT = false, bW = false, bS = false, bB = false;

		if (iE >= t.maxelm) bE = true;
		if (iN >= t.maxelm) bN = true;
		if (iT >= t.maxelm) bT = true;
		if (iW >= t.maxelm) bW = true;
		if (iS >= t.maxelm) bS = true;
		if (iB >= t.maxelm) bB = true;

		if ((bE) || (bW)) {
			dl = 0.5*dx;
			//if (bE) resterm[iE]=0.1*dl*dl*dl; // O(h!3)
			//if (bW) resterm[iW]=0.1*dl*dl*dl; // O(h!3)
			if (bE) resterm[iE] = dl; // O(h)
			if (bW) resterm[iW] = dl; // O(h)

		}
		if ((bN) || (bS)) {
			dl = 0.5*dy;
			//if (bN) resterm[iN]=0.1*dl*dl*dl; // O(h!3)
			//if (bS) resterm[iS]=0.1*dl*dl*dl; // O(h!3)
			if (bN) resterm[iN] = dl; // O(h)
			if (bS) resterm[iS] = dl; // O(h)

		}
		if ((bT) || (bB)) {
			dl = 0.5*dz;
			//if (bT) resterm[iT]=0.1*dl*dl*dl; // O(h!3)
			//if (bB) resterm[iB]=0.1*dl*dl*dl; // O(h!3)
			if (bT) resterm[iT] = dl; // O(h)
			if (bB) resterm[iB] = dl; // O(h)
		}
	}
	doublereal ret;
	//ret=Scal(resterm,resterm,f.maxelm+f.maxbound);
	ret = NormaV(resterm, t.maxelm + t.maxbound);
	// ������������ ����������� ������.
	if (resterm != NULL) {
		delete[] resterm;
		resterm = NULL;
	}
	return ret;
} // rterminate_residual_LR1sk_temp_Oh3



int main(void)
{
	getchar();



	

	// 22.01.2017 �������������.
	eqin.fluidinfo = NULL;
	t.rootBT = NULL;
	t.rootSN = NULL;
	t.rootWE = NULL;

	// 29 10 2016.
	// ������������� ����� ������ � ILU �������.
	milu_gl_buffer.alu_copy = NULL;
	milu_gl_buffer.jlu_copy = NULL;
	milu_gl_buffer.ju_copy = NULL;
	
	//fgmres(m_restart)
	my_amg_manager.m_restart = 20; // ���������� �������� ��������� fgmres ����� ������������.

	// amg default settings:
	my_amg_manager.lfil = 2; // default value

	// ��������� ������������ �������������� ������ � ���������.
	// ��������� �������� ���� ������� �� ���� ��������� ������� ������� �� ����:
	// �������������� <-> ��������������, ����������������, ������������ ������ <-> ������������ ������,
	// ������� ������������ ��������� (����� ����������).
	//my_amg_manager.maximum_levels = 20; // ������������ ����� ������� ������� � �������� ���������� ��������.
	my_amg_manager.maximum_delete_levels = 0; // ���������� ������� ���������� � ������ ����� ��� ������ �����.
	my_amg_manager.number_interpolation_procedure = 3; // ����� ���������������� ���������.
	my_amg_manager.number_interpolation_procedure_Temperature = 3;
	my_amg_manager.number_interpolation_procedure_Speed = 3;
	my_amg_manager.number_interpolation_procedure_Pressure = 3;
	my_amg_manager.number_interpolation_procedure_Stress = 3;

	my_amg_manager.iCFalgorithm_and_data_structure=3; // 3-Treap.
	my_amg_manager.iCFalgorithm_and_data_structure_Temperature=3;// 3-Treap.
	my_amg_manager.iCFalgorithm_and_data_structure_Speed=3;// 3-Treap.
	my_amg_manager.iCFalgorithm_and_data_structure_Pressure=3;// 3-Treap.
	my_amg_manager.iCFalgorithm_and_data_structure_Stress=3;// 3-Treap.

	my_amg_manager.bTemperatureMatrixPortrait = 0; // NO_PRINT
	my_amg_manager.bSpeedMatrixPortrait = 0; // NO_PRINT
	my_amg_manager.bPressureMatrixPortrait = 0; // NO_PRINT
	my_amg_manager.bStressMatrixPortrait = 0; // NO_PRINT
	my_amg_manager.bMatrixPortrait = 0; // NO_PRINT


	my_amg_manager.nFinnest = 2; // ����� �������� �� ��������� �����.
	my_amg_manager.nu1 = 1; // ����� ���������������.
	my_amg_manager.nu2 = 2; // ����� ���� ����������.	
	my_amg_manager.memory_size = 9; // ���������� ����������� ������ � ������������ ������� �.
	my_amg_manager.gold_const = 0.2; // �������� ������� ���������� � ������������.
	my_amg_manager.gold_const_Temperature = 0.2;
	my_amg_manager.gold_const_Speed = 0.2;
	my_amg_manager.gold_const_Pressure = 0.2;
	my_amg_manager.gold_const_Stress = 0.2;
	my_amg_manager.bCFJacoby = true; // CF-Jacobi smoothers 12% ���������� ����� V ������. 5.06.2017
	// Runge-Kutt smoother: 3 - �������� �������, 5 - ������ �������, ����� ������ ����� �� ������������.
	my_amg_manager.iRunge_Kutta_smoother = 0;
	my_amg_manager.iFinnest_ilu = 0; // 0 - �� ������������, 1 - ������������, �� ������ �� ����� ��������� �����.
	// ������������� iluk ���������� �� �������� ������� ����������� ��� �������
	// �������� ������ nnz/n ����� ������ ������ ���� ������ 6 (�����).
	my_amg_manager.b_ilu_smoothers_in_nnz_n_LE_6 = false;
	my_amg_manager.theta = 0.24;
	my_amg_manager.magic = 0.4;
	my_amg_manager.F_to_F_Temperature = 0.4;
	my_amg_manager.F_to_F_Speed = 0.4; 
	my_amg_manager.F_to_F_Pressure = 0.4;
	my_amg_manager.F_to_F_Stress = 0.4;
	my_amg_manager.ilu2_smoother = 0; // 0 - �� ������������, 1 - ������������.

	my_amg_manager.itypemodifyinterpol=0; // ����� ����������� �������������.
	my_amg_manager.inumberadaptpass=0; // ������������ ���������� ������-�������� � �������������.

	my_amg_manager.theta_Temperature = 0.24;
	my_amg_manager.maximum_delete_levels_Temperature = 0;
	my_amg_manager.nFinnest_Temperature = 2;
	my_amg_manager.nu1_Temperature = 1;
	my_amg_manager.nu2_Temperature = 2;
	my_amg_manager.memory_size_Temperature = 9;
	my_amg_manager.ilu2_smoother_Temperature = 0; // 0 - �� ������������, 1 - ������������.
	// Speed
	my_amg_manager.theta_Speed = 0.24;
	my_amg_manager.maximum_delete_levels_Speed = 0;
	my_amg_manager.nFinnest_Speed = 2;
	my_amg_manager.nu1_Speed = 1;
	my_amg_manager.nu2_Speed = 2;
	my_amg_manager.memory_size_Speed = 9;
	my_amg_manager.ilu2_smoother_Speed = 0; // 0 - �� ������������, 1 - ������������.
	// Pressure
	my_amg_manager.theta_Pressure = 0.24;
	my_amg_manager.maximum_delete_levels_Pressure = 0;
	my_amg_manager.nFinnest_Pressure = 2;
	my_amg_manager.nu1_Pressure = 1;
	my_amg_manager.nu2_Pressure = 2;
	my_amg_manager.memory_size_Pressure = 9;
	my_amg_manager.ilu2_smoother_Pressure = 0; // 0 - �� ������������, 1 - ������������.
	// Stress
	my_amg_manager.theta_Stress = 0.24;
	my_amg_manager.maximum_delete_levels_Stress = 0;
	my_amg_manager.nFinnest_Stress = 2;
	my_amg_manager.nu1_Stress = 1;
	my_amg_manager.nu2_Stress = 2;
	my_amg_manager.memory_size_Stress = 9;
	my_amg_manager.ilu2_smoother_Stress = 0; // 0 - �� ������������, 1 - ������������.
	// AMG Splitting (coarsening)
	// ������ ���������� C-F ��������� : 0 - standart, 1 - RS 2.
	// RS 2 ���������� ������ ���������� C-F ��������� ���������� ������ ������.
	my_amg_manager.icoarseningTemp = 0; // standart
	my_amg_manager.icoarseningSpeed = 0; // standart
	my_amg_manager.icoarseningPressure=0; // standart
	my_amg_manager.icoarseningStress = 0; // standart
	my_amg_manager.icoarseningtype=0; // standart vs RS 2.
	// Stabilization BiCGStab.
	// 8.01.2017 ����� ��� ��� ������ BiCGStab 
	// ����������������� ��������������� ������������� �������.
	// 0 - ������������ ������ �������������� ������������� ����� ��� ������-���� ����������� ���������� ��������������� �������,
	// 1 - ������������ �������� �. ��� ��� ������ BiCGStab [1992], ����������������� �������������� ������������� �������.
	// 2 - ������������ �������� ����� � ������ FGMRes [1986], ����������������� �������������� ������������� �������.
	my_amg_manager.istabilizationTemp = 0; // none
	my_amg_manager.istabilizationSpeed = 0; // none
	my_amg_manager.istabilizationPressure = 0; // none
	my_amg_manager.istabilizationStress = 0; // none
	my_amg_manager.istabilization = 0; // none

	// ����� ������������ �����.
	my_amg_manager.ipatch_number = 0; // 0 - ���� �� ������������.

	// ������ ���� �� �������.
	my_amg_manager.iprint_log = 1;
	my_amg_manager.iprint_log_Temperature = 1;
	my_amg_manager.iprint_log_Speed = 1;
	my_amg_manager.iprint_log_Pressure=1;
	my_amg_manager.iprint_log_Stress = 1;

	// truncation for interpolation.
	// �� ��������� �������� ������������ �� ������������.
	my_amg_manager.itruncation_interpolation = 0; // 0 - off
	my_amg_manager.itruncation_interpolation_Temperature = 0;
	my_amg_manager.itruncation_interpolation_Speed = 0;
	my_amg_manager.itruncation_interpolation_Pressure=0;
	my_amg_manager.itruncation_interpolation_Stress = 0;
	// 0.2 recomended Stuben.
	my_amg_manager.truncation_interpolation = 0.2; // 0.2 recomended default value.
	my_amg_manager.truncation_interpolation_Temperature = 0.2;
	my_amg_manager.truncation_interpolation_Speed = 0.2;
	my_amg_manager.truncation_interpolation_Pressure=0.2;
	my_amg_manager.truncation_interpolation_Stress = 0.2;

	// GMRES smoother.
	my_amg_manager.b_gmresTemp = false;
	my_amg_manager.b_gmresSpeed = false;
	my_amg_manager.b_gmresPressure=false;
	my_amg_manager.b_gmresStress = false;
	my_amg_manager.b_gmres=false;

	// ����� �������.
	unsigned int calculation_main_start_time = 0; // ������ ����� ��.
	unsigned int calculation_main_end_time = 0; // ��������� ����� ��.
	unsigned int calculation_main_seach_time = 0; // ����� ���������� ������� ���� � ��.

	calculation_main_start_time = clock(); // ������ ������ �����.

	bool bextendedprint = false; // ������ �� ��������� ����� ����������� �����.

	// ��������������� ������ ��� �������� ���������� ����������.
	err_radiation_log = fopen_s(&fp_radiation_log, "log_radiation.txt", "a");
	if (err_radiation_log != 0) {
		printf("Error open file log.txt\n");
		printf("Please, press any key to continue...\n");
		//getchar();
		system("pause");
		exit(0);
	}

	//std::locale::global(std::locale("en_US.UTF-8"));
	system("mode con cols=126 lines=12000");
	// � ������� ��� ���� ����� �������� ���� ������ � ������� ����� ������� � ������� ���
	//HANDLE hOCol = GetStdHandle(STD_OUTPUT_HANDLE);
	//SetConsoleTextAttribute(hOCol, FOREGROUND_GREEN);
	// ��������� ��������� ������� (����� ���)	
	//SetConsoleTextAttribute(hOCol, BACKGROUND_BLUE |
	//	BACKGROUND_GREEN |
	//	BACKGROUND_RED |
	//	BACKGROUND_INTENSITY);

	//system("cls");
	// ���������� � ���, ��� ����� ��� ���������� ���� ��� ��� ��������� �����.
		

	printf("AliceFlow 3D x64 v0_07\n");
#ifdef _OPENMP 
	omp_set_num_threads(inumcore); // ��������� ����� �������
#endif

	errno_t err;
	err = fopen_s(&fp_log, "log.txt", "w");
	if (err != 0) {
		printf("Error open file log.txt\n");
		printf("Please, press any key to continue...\n");
		//getchar();
		system("pause");
		exit(0);
	}

	if (fp_log != NULL) {

		//ilu0_Saadtest();
		//printf("the end Saad ilu0 test\n");
		//getchar();

		// ���������� ����� �� ������ �� ����.
		//integer inx=120, iny=64, inz=64;
		integer inx = 30, iny = 30, inz = 30;
		integer inxadd = -1, inyadd = -1, inzadd = -1;
		doublereal dgx = 0.0, dgy = 0.0, dgz = 0.0; // ���� �������
		doublereal operatingtemperature = 20.0; // Operating Temperature 20.0 ����. �.

		

		premeshin("premeshin.txt", lmatmax, lb, ls, lw, matlist, b, s, w, dgx, dgy, dgz, inx, iny, inz, operatingtemperature,  ltdp, gtdps);
		if (iswitchMeshGenerator == 0) {
			simplemeshgen(xpos, ypos, zpos, inx, iny, inz, lb, ls, lw, b, s, w, matlist, xposadd, yposadd, zposadd, inxadd, inyadd, inzadd);
		}
		else if (iswitchMeshGenerator == 1) {
			unevensimplemeshgen(xpos, ypos, zpos, inx, iny, inz, lb, ls, lw, b, s, w, matlist, dgx, dgy, dgz, xposadd, yposadd, zposadd, inxadd, inyadd, inzadd); // ��������� ������������� ����� � �������������� ������������� ���������������.
		}
		else if (iswitchMeshGenerator == 2) {
			// � ��������� ������� coarse Mesh ��� � Icepak.
			// �������� ������ ����������� ��������������-���������������, � 
			// ������� ������������� ���������� ����������� �����, �.�. cpu �������� � 4��� �
			// ���������� ����������������� ��������� ������� ������� ��������.
			coarsemeshgen(xpos, ypos, zpos, inx, iny, inz, lb, ls, lw, b, s, w, matlist, xposadd, yposadd, zposadd, inxadd, inyadd, inzadd);
		}
		else {
#if doubleintprecision == 1
			printf("error : yuor mesh generator is undefined %lld\n", iswitchMeshGenerator);
#else
			printf("error : yuor mesh generator is undefined %d\n", iswitchMeshGenerator);
#endif
			
			system("pause");
			exit(1);
		}


		if (b_on_adaptive_local_refinement_mesh) {
			printf("starting ALICE\n");
			// ��� ������� ������� �������� �� ������ ������������ �������,
			// � ������ �������� �� ��������� ���� � ��������� new ��� malloc.
			integer maxelm_loc = (inx + 1)*(iny + 1)*(inz + 1);
			bool bOkal=alice_mesh(xpos, ypos, zpos, inx, iny, inz, b, lb, lw, w, s, ls, maxelm_loc, xposadd, yposadd, zposadd, inxadd, inyadd, inzadd);
			//system("PAUSE");
			
			if (itype_ALICE_Mesh == 1) {
				// �������� ��������� ���������.

				while (!bOkal) {
					/* 3.09.2017
					���� ����� ����� ������� ����� ��� ��������� ������ ��������������� �������������� ����� �������
					����� �������. ������ ������� ������� ������������� ������ � ���������� ���� �� ����� ����������� �������������
					����� ���� ������ ���� ����� ������ ���������. � ���������� �������� ������������� ����������� ��������. ������ �����
					��������� ������� ������ �������������� ����� ����� ��������� ����� ����� ��� ������������ ����� (�.�. ����� ����������
					��� ���������� ����� �������� ����� ������� ����� ������ �������� ����� ���������� �� ����� ��� �� 1. ������� ������
					����� ����������� ��� ��������� ������ ������ � ������� if_disbalnce(...) � ��� ������ ����� ������ ����������� �������
					�������� ��������������� ����� �������� �����. �������� ����������� ������ � ����������� �� �������� ������� � ����������
					���� ����� ���������� ������ ������ ������ ������� ����� ��� �������� ����������� �������� �����.
					*/


					// ����� ���������� ������ �� ��� octree ������ � ����������� �����.
					printf("free octree start...\n");
					//getchar();
					//system("PAUSE");
					free_octree(oc_global);
					delete[] my_ALICE_STACK;
					top_ALICE_STACK = 0;
					printf("free octree end...\n");
					// ����� ���������� ��������� �����.
					delete[] xpos;
					xpos = NULL;
					inx = 0;
					delete[] ypos;
					ypos = NULL;
					iny = 0;
					delete[] zpos;
					zpos = NULL;
					inz = 0;

					printf("free xpos, ypos, zpos\n");
					//system("PAUSE");

					if (iswitchMeshGenerator == 0) {
						simplemeshgen(xpos, ypos, zpos, inx, iny, inz, lb, ls, lw, b, s, w, matlist, xposadd, yposadd, zposadd, inxadd, inyadd, inzadd);
					}
					else if (iswitchMeshGenerator == 1) {
						unevensimplemeshgen(xpos, ypos, zpos, inx, iny, inz, lb, ls, lw, b, s, w, matlist, dgx, dgy, dgz, xposadd, yposadd, zposadd, inxadd, inyadd, inzadd); // ��������� ������������� ����� � �������������� ������������� ���������������.
					}
					else if (iswitchMeshGenerator == 2) {
						// � ��������� ������� coarse Mesh ��� � Icepak.
						// �������� ������ ����������� ��������������-���������������, � 
						// ������� ������������� ���������� ����������� �����, �.�. cpu �������� � 4��� �
						// ���������� ����������������� ��������� ������� ������� ��������.
						coarsemeshgen(xpos, ypos, zpos, inx, iny, inz, lb, ls, lw, b, s, w, matlist, xposadd, yposadd, zposadd, inxadd, inyadd, inzadd);
					}
					else {
#if doubleintprecision == 1
						printf("error : yuor mesh generator is undefined %lld\n", iswitchMeshGenerator);
#else
						printf("error : yuor mesh generator is undefined %d\n", iswitchMeshGenerator);
#endif

						system("pause");
						exit(1);
					}
					// ����� ������ ���� �����������.

					printf("new construct xpos, ypos, zpos\n");
					//system("PAUSE");

					bOkal = alice_mesh(xpos, ypos, zpos, inx, iny, inz, b, lb, lw, w, s, ls, maxelm_loc, xposadd, yposadd, zposadd, inxadd, inyadd, inzadd);

					//getchar();
					//system("PAUSE");
				}
			}
			printf("end ALICE\n");
		}

		load_TEMPER_and_FLOW(t, f, inx, iny, inz, xpos, ypos, zpos, flow_interior, b, lb, lw, w, s, ls, operatingtemperature, matlist, bextendedprint, dgx, dgy, dgz, b_on_adaptive_local_refinement_mesh, false);
		
		// ��� ����� ������ ����� ��� ������� �������������� ������.
		t.inx_copy = inx;
		t.iny_copy = iny;
		t.inz_copy = inz;
		t.operatingtemperature_copy = operatingtemperature;
		t.xpos_copy = new doublereal[inx + 1];
		t.ypos_copy = new doublereal[iny + 1];
		t.zpos_copy = new doublereal[inz + 1];
		// ������ ������������� ����� ��� �������� ����������� ������,
		// ��������� ������ ����� ��������� �� ��� � ����� ������������� 
		// ���� ���������.
		for (integer i_7 = 0; i_7 < inx + 1; i_7++) {
			t.xpos_copy[i_7] = xpos[i_7];
		}
		for (integer i_7 = 0; i_7 < iny + 1; i_7++) {
			t.ypos_copy[i_7] = ypos[i_7];
		}
		for (integer i_7 = 0; i_7 < inz + 1; i_7++) {
			t.zpos_copy[i_7] = zpos[i_7];
		}

		// ������������ ����������� ������ �� ��� octree ������.
		if (b_on_adaptive_local_refinement_mesh) {
			printf("free octree start...\n");
			//getchar();
			//system("PAUSE");
			free_octree(oc_global);
			delete[] my_ALICE_STACK;
			top_ALICE_STACK = 0;
			printf("free octree end...\n");
			//getchar();
			//system("PAUSE");
		}

		if (0) {
			xyplot(f, flow_interior, t);
			printf("after load temper and flow. OK.\n");
			//getchar(); // debug avtosave
			system("pause");
		}

		// �� ���� ����� ���������� ����� �� �������� ���������������.
		if (!b_on_adaptive_local_refinement_mesh) {
			// ������ ���������� � �������� ������ ��� ������������� ������ LR:
			constr_line(f, flow_interior);  // ��� �������������
			t.rootBT = NULL;
			t.rootSN = NULL;
			t.rootWE = NULL;
			constr_line_temp(t, b, lb); // ��� ����������������
			printf("LR preprocessing finish...\n");
		}

		// ���������� ������ ��� ��������������� �������������� ������.

		amgGM.a = NULL;
		amgGM.f = NULL;
		amgGM.ia = NULL;
		amgGM.ig = NULL;
		amgGM.ja = NULL;
		amgGM.u = NULL;
		amgGM.nda = -1;
		amgGM.ndf = -1;
		amgGM.ndia = -1;
		amgGM.ndig = -1;
		amgGM.ndja = -1;
		amgGM.ndu = -1;


		//PARDATA nd;
		nd.ncore = 2; // ��� ����.
		// �� ��������� ��� ��������� ���������.
		nd.b0.active = false;
		nd.b00.active = false;
		nd.b01.active = false;
		nd.b000.active = false;
		nd.b001.active = false;
		nd.b010.active = false;
		nd.b011.active = false;
		if (0 && (flow_interior == 1)) {
			calc_front(f, f[0], t, flow_interior, ls, lw, w, nd);
			// ���������� ��������� !
			printf("separator compleate...\n");
			//getchar();
		}



		t.free_temper_level1 = false; // ������ ���������������� ������������ ������ ����������� ��� ������ ������� ����� �������� ������.
		t.free_temper_level2 = false; // ������������ ������ ��� �������� ������� ��� ���������� � � SIMPLESPARSE ������.	

		printf("construction of all structures...\n");
		printf("mesh check start...\n");
#if doubleintprecision == 1
		for (integer i = 0; i < inx; i++) if (fabs(xpos[i + 1] - xpos[i]) < 1.0e-23)
			printf("error: zalipanie po X: xpos[%lld]=%e xpos[%lld]=%e inx=%lld\n", i, xpos[i], i + 1, xpos[i + 1], inx);
		for (integer i = 0; i < iny; i++) if (fabs(ypos[i + 1] - ypos[i]) < 1.0e-23)
			printf("error: zalipanie po X: ypos[%lld]=%e ypos[%lld]=%e iny=%lld\n", i, ypos[i], i + 1, ypos[i + 1], iny);
		for (integer i = 0; i < inz; i++) if (fabs(zpos[i + 1] - zpos[i]) < 1.0e-23)
			printf("error: zalipanie po X: zpos[%lld]=%e zpos[%lld]=%e inz=%lld\n", i, zpos[i], i + 1, zpos[i + 1], inz);
		for (integer iP = 0; iP < t.maxelm; iP++) {
			if ((t.nvtx[0][iP] == 0) || (t.nvtx[1][iP] == 0) || (t.nvtx[2][iP] == 0) || (t.nvtx[3][iP] == 0) || (t.nvtx[4][iP] == 0) || (t.nvtx[5][iP] == 0) || (t.nvtx[6][iP] == 0) || (t.nvtx[7][iP] == 0)) {
				printf("nvtx[%lld] : %lld %lld %lld %lld %lld %lld %lld %lld \n", iP, t.nvtx[0][iP] - 1, t.nvtx[1][iP] - 1, t.nvtx[2][iP] - 1, t.nvtx[3][iP] - 1, t.nvtx[4][iP] - 1, t.nvtx[5][iP] - 1, t.nvtx[6][iP] - 1, t.nvtx[7][iP] - 1);
			}
		}
#else
		for (integer i = 0; i < inx; i++) if (fabs(xpos[i + 1] - xpos[i]) < 1.0e-23)
			printf("error: zalipanie po X: xpos[%d]=%e xpos[%d]=%e inx=%d\n", i, xpos[i], i + 1, xpos[i + 1], inx);
		for (integer i = 0; i < iny; i++) if (fabs(ypos[i + 1] - ypos[i]) < 1.0e-23)
			printf("error: zalipanie po X: ypos[%d]=%e ypos[%d]=%e iny=%d\n", i, ypos[i], i + 1, ypos[i + 1], iny);
		for (integer i = 0; i < inz; i++) if (fabs(zpos[i + 1] - zpos[i]) < 1.0e-23)
			printf("error: zalipanie po X: zpos[%d]=%e zpos[%d]=%e inz=%d\n", i, zpos[i], i + 1, zpos[i + 1], inz);
		for (integer iP = 0; iP < t.maxelm; iP++) {
			if ((t.nvtx[0][iP] == 0) || (t.nvtx[1][iP] == 0) || (t.nvtx[2][iP] == 0) || (t.nvtx[3][iP] == 0) || (t.nvtx[4][iP] == 0) || (t.nvtx[5][iP] == 0) || (t.nvtx[6][iP] == 0) || (t.nvtx[7][iP] == 0)) {
				printf("nvtx[%d] : %d %d %d %d %d %d %d %d \n", iP, t.nvtx[0][iP] - 1, t.nvtx[1][iP] - 1, t.nvtx[2][iP] - 1, t.nvtx[3][iP] - 1, t.nvtx[4][iP] - 1, t.nvtx[5][iP] - 1, t.nvtx[6][iP] - 1, t.nvtx[7][iP] - 1);
			}
		}
#endif
		
		
		// �� ����� ������ ������ ���� � ��������� ����������� ������� �������������.
		// ������� ������������� O(h!2) ������� ������� ������������ ��������� ������������ ������,
		// �.�. ������� �� ����������� ��������� �����.
		for (integer i = 0; i < flow_interior; i++) {
#if doubleintprecision == 1
			printf("FLUID %lld\n", i);
#else
			printf("FLUID %d\n", i);
#endif
			
			// �������� � ������� ���������������� ��������� ��� �������� ��������.
			f[i].resICCG = rterminate_residual_ICCG_Oh2(f[i]); // O(h!2)
			printf("residual O(h!2) is equal=%e\n", f[i].resICCG);
			f[i].resLR1sk = rterminate_residual_LR1sk_Oh3(f[i]); // O(h!3)
			printf("residual O(h!3) is equal=%e\n", f[i].resLR1sk);
		}
		printf("TEMPERATURE\n");
		t.resLR1sk = rterminate_residual_LR1sk_temp_Oh3(t); // O(h!3)		
		printf("temp residual O(h!3) is equal=%e\n", t.resLR1sk);
		printf("mesh check.\n");
		if (bwait) {
			//getchar();
			system("pause");
		}
		
		sourse2Dproblem = new bool[t.maxbound];
		conductivity2Dinsource = new doublereal[t.maxbound];
		// ������ ���������� ������������� ����� ��� ����������� �������.
		bsource_term_radiation_for_relax = new doublereal[t.maxelm];
		for (integer i_init = 0; i_init < t.maxelm; i_init++) bsource_term_radiation_for_relax[i_init] = 0.0;
		b_buffer_correct_source = new doublereal[t.maxelm];


		// ������� continity ����� ���������� �� ��������� � ������ 1e0.
		doublereal* continity_start = NULL;
		continity_start = new doublereal[flow_interior];
		if (continity_start == NULL) {
			// ������������ ������ �� ������ ������������.
			printf("Problem : not enough memory on your equipment for continity start in main...\n");
			printf("Please any key to exit...\n");
			exit(1);
		}
		for (integer i = 0; i < flow_interior; i++) continity_start[i] = 1.0;

		integer* inumber_iteration_SIMPLE = NULL;
		inumber_iteration_SIMPLE = new integer[flow_interior];
		if (inumber_iteration_SIMPLE == NULL) {
			// ������������ ������ �� ������ ������������.
			printf("Problem : not enough memory on your equipment for inumber_iteration_SIMPLE in main...\n");
			printf("Please any key to exit...\n");
			exit(1);
		}
		for (integer i = 0; i < flow_interior; i++) inumber_iteration_SIMPLE[i] = 0; // ��������� �������� ��������� SIMPLE ��� ������ FLUID ����.

		// ���������� ��������� ������� �� ����� ��� ������������� �������
		bool breadOk = false;
		avtoreadvalue(f, t, flow_interior, inumber_iteration_SIMPLE, continity_start, breadOk);
		// ���� ���������� ������ ��������� �� breadOk==false � ��� ������ ��� ���� ������� ������ �� �������� �������� ��� �������������.

		if (b_on_adaptive_local_refinement_mesh) {
			// ��������� ������������ ���� �����.
			printf("the invariant correctness...\n");
			ANES_ALICE_CORRECT(t.maxnod, t.pa, t.maxelm, t.nvtx);
		}

		// ������� ���������� ���������� � ��������� tecplot360:
		// ����� ������������ ��� �������� ����������� �����.
		if (0) {
			exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint,0);
			printf("read values. OK.\n");
			//getchar(); // debug avtosave
			system("pause");
		}



		/*for (integer i=0; i<lw; i++) {
		printf("%e  \n",w[i].Tamb);
		}
		//exporttecplotxy360T_3D(t.maxelm, t.ncell, t.nvtx, t.nvtxcell, t.pa, t.potent);
		exporttecplotxy360T_3D_part2(t.maxelm, t.potent);
		getchar(); // debug
		*/

		// 29.01.2017
		// if (1 && steady_or_unsteady_global_determinant == 2)  
		// �� �� ������ �������� ����� �� ������� �������.
		if (1 && steady_or_unsteady_global_determinant == 2) {
			// ����� �������.
			unsigned int calculation_start_time = 0; // ������ ����� ��.
			unsigned int calculation_end_time = 0; // ��������� ����� ��.
			unsigned int calculation_seach_time = 0; // ����� ���������� ������� ���� � ��.

			calculation_start_time = clock(); // ������ ������ �����.

			// ���������� ����� ������.
			massa_cabinet(t, f, inx, iny, inz,
				xpos, ypos, zpos, flow_interior,
				b, lb, operatingtemperature,
				matlist);

			calculation_end_time = clock(); // ������ ��������� �����.
			calculation_seach_time = calculation_end_time - calculation_start_time;
			unsigned int im = 0, is = 0, ims = 0;
			im = (unsigned int)(calculation_seach_time / 60000); // ������
			is = (unsigned int)((calculation_seach_time - 60000 * im) / 1000); // �������
			ims = (unsigned int)((calculation_seach_time - 60000 * im - 1000 * is) / 10); // ������������ ������� �� 10

			printf("time export to tecplot360 is:  %d minute %d second %d millisecond\n", im, is, 10 * ims);

			// ������� ����� � tecplot 360.
			if (1) {
				if (!b_on_adaptive_local_refinement_mesh) {
					// ������� ���������� ���������� � ��������� tecplot360:
					exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint, 0);
				}
				else {
					// ������� � ��������� ������� �����������.
					//� ���� �����.
					ANES_tecplot360_export_temperature(t.maxnod, t.pa, t.maxelm, t.nvtx, t.potent, t,0);
				}
			}

			
		}

		// steady
		if (1 && steady_or_unsteady_global_determinant == 0) {

			// ����� �������.
			unsigned int calculation_start_time = 0; // ������ ����� ��.
			unsigned int calculation_end_time = 0; // ��������� ����� ��.
			unsigned int calculation_seach_time = 0; // ����� ���������� ������� ���� � ��.

			calculation_start_time = clock(); // ������ ������ �����.

			for (integer i7 = 0; i7<t.maxelm + t.maxbound; i7++) t.potent[i7] = operating_temperature_for_film_coeff; // �������������.

			// ������ ������ ������������� � ������ ����.
			bonly_solid_calculation = true;

			// �������� ����������� ���������� �� ����������� ������.
			if (lw == 1) {
				bPhysics_stop = true;
				if (lb < 11) {
					// ��� ����������� �������� :
					// MD40, AuSn, Cu, AuSn, SiC, GaN. cabinet and hollow.
					bPhysics_PTBSH_memory = true;
				}
			}

			if (adiabatic_vs_heat_transfer_coeff == 1) {
				// �� �������������� ��������� ��������� ����� ��������� �� ��������������� ��� ������ ������ ����.
				//for (integer i7 = 0; i7<t.maxelm + t.maxbound; i7++) t.potent[i7] = 0.57*operating_temperature_for_film_coeff;
			}

			// ����� �������������� ��� �� ������ ������������ ������ ������ ����������������.
			bsolid_static_only = true;
			bool bcleantemp = false;
			if (eqin.itemper == 1) {
				bcleantemp = true;
				integer i = 0; // ������� �����
				for (i = 0; i < flow_interior; i++) {
					if (eqin.fluidinfo[i].iflow == 1) bcleantemp = false;
				}
				// ���� bcleantemp==true �� �� ������ ������ ������ ������������� ��� ����� ���������.
			}
			if (1 || bcleantemp) {
				// ������� ������������ ���������� (��� ��������) ������ ������ ���������������� � ��������� �������. 
				printf("solution of pure heat...\n");
				printf("please, press any key to continue...\n");
				if (bwait) {
					//getchar();
					system("pause");
				}

				// ��� ������������ ������������� ����������� ��������.
				bool bprintmessage = true; // �������� �� ��������� �� �������.

				doublereal dbeta = 1.0; // ������ ������� ������������� �� �������.
				bool bmyconvective = false;
				if (starting_speed_Vx*starting_speed_Vx + starting_speed_Vy*starting_speed_Vy + starting_speed_Vz*starting_speed_Vz > 1.0e-30) {
					bmyconvective = true;
				}
				else {
					// �������� ������������� ��������� ��������.
					errno_t err_inicialization_data;
					FILE* fp_inicialization_data;
					err_inicialization_data = fopen_s(&fp_inicialization_data, "load.txt", "r");
					if (err_inicialization_data == 0) {
						// �������� ������ � ���� ������������.
						bmyconvective = true;
						fclose(fp_inicialization_data);
					}
				}

				// if (flow_interior>0) bmyconvective=true;
				// ������ ���������� ����������,
				// ��������� ��� �������� ������� ���-���
				doublereal **rhie_chow = NULL;
				QuickMemVorst m;
				m.ballocCRSt = false; // �������� ������
				m.bsignalfreeCRSt = true; // � ����� �����������.
				// ������������� ����������.
				m.tval = NULL;
				m.tcol_ind = NULL;
				m.trow_ptr = NULL;
				m.tri = NULL;
				m.troc = NULL;
				m.ts = NULL;
				m.tt = NULL;
				m.tvi = NULL;
				m.tpi = NULL;
				m.tdx = NULL;
				m.tdax = NULL;
				m.ty = NULL;
				m.tz = NULL;
				m.ta = NULL;
				m.tja = NULL;
				m.tia = NULL;
				m.talu = NULL;
				m.tjlu = NULL;
				m.tju = NULL;
				m.tiw = NULL;
				m.tlevs = NULL;
				m.tw = NULL;
				m.tjw = NULL;
				m.icount_vel = 100000; // ����� ������� �����.			
				
				
				
				// ���� flow_interior == 0 �� f[0] ������ ���������� ��������  
				solve_nonlinear_temp(f[0], f, t,
					rhie_chow,
					b, lb, s, ls, w, lw,
					dbeta, flow_interior,
					bmyconvective, NULL, 0.001, 0.001,
					false,
					matlist, 0,
					bprintmessage,
					gtdps, ltdp, 1.0, m,
					NULL, // �������� � ����������� ���������� ����. 
					NULL); // �������� ����� ����� ������� � ����������� ���������� ����.
				// ��������� �������� ������ 1.0 �������� ��� �������� �������.
				
				// ���������� ����� ������.
				massa_cabinet(t, f, inx, iny, inz,
					xpos, ypos, zpos, flow_interior,
					b, lb, operatingtemperature,
					matlist);

				// 10.10.2017
				// ���������� ���������� ������� ����� ���������.
				xyplot_temp(t, t.potent);
				//printf("graphics writing sucseful\n");
				//getchar();

				if (1) {
					if (!b_on_adaptive_local_refinement_mesh) {
						// ������� ���������� ���������� � ��������� tecplot360:
						exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint,0);
					}
					else {
						// ������� � ��������� ������� �����������.
						//� ���� �����.
						//ANES_tecplot360_export_temperature(t.maxnod, t.pa, t.maxelm, t.nvtx, t.potent,t,0);
						
					}
				}
				//printf("marker stop\n");
				//getchar();
			}

			doublereal tmaxfinish = -273.15; // ���������� ����.
			// ���������� �������� ������������ ����������� ������ ��������� ������� � �� � ��������:
			//for (integer i = 0; i < t.maxelm + t.maxbound; i++) tmaxfinish = fmax(tmaxfinish, fabs(t.potent[i]));
			// 23 ������� 2015
			// �� ��������� ������ ���������� ����� �� ����� ��������� ������� �����������, �������
			// �������� �� ������� ����� � ��������� ����������� ������ �� ���������� ��. 
			for (integer i = 0; i < t.maxelm; i++) tmaxfinish = fmax(tmaxfinish, t.potent[i]);
			FILE *fp;
			errno_t err1;
			err1 = fopen_s(&fp, "report.txt", "w");
			// �������� ����� ��� ������.
			if ((err1) != 0) {
				printf("Create File report.txt Error\n");
				//getchar();
				system("pause");
			}
			else {
				// ������ ���������
				fprintf(fp, "Maximum Temperature %.2f\n", tmaxfinish);
				fclose(fp);
			}
			// 1 - solver/solid_static/
			report_temperature(flow_interior, f, t, b, lb, s, ls, w, lw, 0);

			if (1) {

				calculation_end_time = clock(); // ������ ��������� �����.
				calculation_seach_time = calculation_end_time - calculation_start_time;
				unsigned int im = 0, is = 0, ims = 0;
				im = (unsigned int)(calculation_seach_time / 60000); // ������
				is = (unsigned int)((calculation_seach_time - 60000 * im) / 1000); // �������
				ims = (unsigned int)((calculation_seach_time - 60000 * im - 1000 * is) / 10); // ������������ ������� �� 10

				printf("time calculation is:  %d minute %d second %d millisecond\n", im, is, 10 * ims);

			    // �� �������� �������������� ������ (11.4� ����� �� ���� Coarse )
				// ����������� ����������� �����. 

				// 25.11.2017
				// 1. �������� ������ � ����������� � ����� �� ����.
				// 2. ��������� �� � ����������� ������.
				// 3. ���������� ������.
				// 4. ��������� ������� ���������� ������������� �����.
				// 5. ��������� ������ � ����������� � ���� �� ������� ���������� ������������� �����.
				// 6. ��������������� ����������� � ����������� �� ����������������� ����� �������� ������.
				// 7.1 ������� �������� ������������� �� ��� ���� �� ����������������� ����� � � ������� � � ������. 
				// 7.2 ������������ �� ���� ����� � ������ ���� �������� ����� ���������� ���� �� ������������������ (���� ��������).


				if (b_on_adaptive_local_refinement_mesh) {
					// 1. ��������� x,y,z,T,nvtx, m_sizeT, m_size_nvtx.
					doublereal *x_buf = NULL;
					doublereal *y_buf = NULL;
					doublereal *z_buf = NULL;
					doublereal *t_buf = NULL;
					integer **nvtx_buf = NULL;
					integer m_sizeT = 0, m_size_nvtx = 0;

					ANES_tecplot360_export_temperature_preobrazovatel(t.maxnod, t.pa, t.maxelm, t.nvtx, t.potent, t, x_buf, y_buf, z_buf, t_buf, nvtx_buf, m_sizeT, m_size_nvtx);

					// 2. ������������ ������.
					// ������������ ����������� ������.
					if (t.xpos_copy != NULL) {
						delete[] t.xpos_copy;
						t.xpos_copy = NULL;
					}
					if (t.ypos_copy != NULL) {
						delete[] t.ypos_copy;
						t.ypos_copy = NULL;
					}
					if (t.zpos_copy != NULL) {
						delete[] t.zpos_copy;
						t.zpos_copy = NULL;
					}


					if (bsource_term_radiation_for_relax != NULL) {
						delete[] bsource_term_radiation_for_relax; // ���������� ������������ ������ ������������ �������.
						bsource_term_radiation_for_relax = NULL;
					}
					if (b_buffer_correct_source != NULL) {
						delete[] b_buffer_correct_source;
						b_buffer_correct_source = NULL;
					}

					if (rthdsd_no_radiosity_patch != NULL) {
						free(rthdsd_no_radiosity_patch);
					}
					rthdsd_no_radiosity_patch = NULL;

					// ������� ��������� ���������� ��������� ������� � ����� 0.14 ��������.
					if (qnbc != NULL) {
						delete[] qnbc;
						qnbc = NULL;
						iadd_qnbc_maxelm = 0;
					}

					// ����� ���������� ����������� ������ �� ��� ���� �������� ������:
					free_level1_temp(t);
					free_level2_temp(t); // ������������ ������ �� ��� ������.
										 // ����������� ������ ��� LR ������.
					if (t.rootWE != NULL) {
						free_root(t.rootWE, t.iWE);
					}
					if (t.rootSN != NULL) {
						free_root(t.rootSN, t.iSN);
					}
					if (t.rootBT != NULL) {
						free_root(t.rootBT, t.iBT);
					}
					if (t.rootWE != NULL) {
						delete[] t.rootWE;
						t.rootWE = NULL;
					}
					if (t.rootSN != NULL) {
						delete[] t.rootSN;
						t.rootSN = NULL;
					}
					if (t.rootBT != NULL) {
						delete[] t.rootBT;
						t.rootBT = NULL;
					}
					// ������������ ������ ��� LR �����.
					free_level1_flow(f, flow_interior);
					free_level2_flow(f, flow_interior); // ������������ ������ �� ��� ������.

					if (sourse2Dproblem != NULL) {
						delete[] sourse2Dproblem;
						sourse2Dproblem = NULL;
					}
					if (conductivity2Dinsource != NULL) {
						delete[] conductivity2Dinsource;
						conductivity2Dinsource = NULL;
					}

					if (x_jacoby_buffer != NULL) {
						// 30 ������� 2016. 
						// � seidelsor2 ������ ������������� �� ����� ������ ���������� �.�. �����.
						// ������������ ������ �� ��� jacobi buffer.
						delete[] x_jacoby_buffer;
					}

					if (bvery_big_memory) {
						if (database.x != NULL) {
							free(database.x);
						}
						if (database.y != NULL) {
							free(database.y);
						}
						if (database.z != NULL) {
							free(database.z);
						}
						if (database.nvtxcell != NULL) {
							for (integer i = 0; i <= 7; i++) {
								delete[] database.nvtxcell[i];
							}
							delete[] database.nvtxcell;
						}
						if (database.ptr != NULL) {
							if (database.ptr[0] != NULL) {
								delete[] database.ptr[0];
							}
							if (database.ptr[1] != NULL) {
								delete[] database.ptr[1];
							}
							delete[] database.ptr;
						}
					}
					/*
					// ������������ ����� ������ � ILU �������.
					if (milu_gl_buffer.alu_copy != NULL) delete[] milu_gl_buffer.alu_copy;
					if (milu_gl_buffer.jlu_copy != NULL) delete[] milu_gl_buffer.jlu_copy;
					if (milu_gl_buffer.ju_copy != NULL) delete[] milu_gl_buffer.ju_copy;
					milu_gl_buffer.alu_copy = NULL;
					milu_gl_buffer.jlu_copy = NULL;
					milu_gl_buffer.ju_copy = NULL;
					*/
					flow_interior = 0;

					// 3. ���������� ������� �����.

					b_on_adaptive_local_refinement_mesh = false;
					load_TEMPER_and_FLOW(t, f, inx, iny, inz, xpos, ypos, zpos, flow_interior, b, lb, lw, w, s, ls, operatingtemperature, matlist, bextendedprint, dgx, dgy, dgz, b_on_adaptive_local_refinement_mesh, false);

					// ��� ����� ������ ����� ��� ������� �������������� ������.
					t.inx_copy = inx;
					t.iny_copy = iny;
					t.inz_copy = inz;
					t.operatingtemperature_copy = operatingtemperature;
					t.xpos_copy = new doublereal[inx + 1];
					t.ypos_copy = new doublereal[iny + 1];
					t.zpos_copy = new doublereal[inz + 1];
					// ������ ������������� ����� ��� �������� ����������� ������,
					// ��������� ������ ����� ��������� �� ��� � ����� ������������� 
					// ���� ���������.
					for (integer i_7 = 0; i_7 < inx + 1; i_7++) {
						t.xpos_copy[i_7] = xpos[i_7];
					}
					for (integer i_7 = 0; i_7 < iny + 1; i_7++) {
						t.ypos_copy[i_7] = ypos[i_7];
					}
					for (integer i_7 = 0; i_7 < inz + 1; i_7++) {
						t.zpos_copy[i_7] = zpos[i_7];
					}

					t.free_temper_level1 = false; // ������ ���������������� ������������ ������ ����������� ��� ������ ������� ����� �������� ������.
					t.free_temper_level2 = false; // ������������ ������ ��� �������� ������� ��� ���������� � � SIMPLESPARSE ������.	


					// 4. ������������� ��� �����������.
					ALICE_2_Structural(t.maxnod, t.pa, t.maxelm, t.nvtx, t.potent, x_buf, y_buf, z_buf, t_buf, nvtx_buf, m_sizeT, m_size_nvtx);


					if (x_buf != NULL) {
						delete[] x_buf;
						x_buf = NULL;
					}
					if (y_buf != NULL) {
						delete[] y_buf;
						y_buf = NULL;
					}
					if (z_buf != NULL) {
						delete[] z_buf;
						z_buf = NULL;
					}
					if (t_buf != NULL) {
						delete[] t_buf;
						t_buf = NULL;
					}
					if (nvtx_buf != NULL) {
						for (integer i_1 = 0; i_1 < 8; i_1++) {
							if (nvtx_buf[i_1] != NULL) {
								delete[] nvtx_buf[i_1];
								nvtx_buf[i_1] = NULL;
							}
						}
						delete[] nvtx_buf;
						nvtx_buf = NULL;
					}
					m_sizeT = 0, m_size_nvtx = 0;
					// 5. ������� ������� � �������.
					exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint, 0);
				}
			}
			else {
				// �������� ������������ �� ���� ����� ������ ���� � ������ �����, �������
				// ��� ������������ �������� ������������ ����� ������� �� ����������������� �����.
				// � ������� �� ���� ����� ��� ������ �� ��������.
				// �������� �� ���� ����� �����. ����� �������� ������� �������� ��� ���� ����������, ��
				// ��������� �������� ������� ������� �� ����� ������� (������������ �����) ������ � ������
				// ��������������� ������ ���� ������ ����� ����� ���� ����� ��������� (���������������� ������).

				// ������� � ��������� ������� �����������.
				// � ���� �����.
				//ANES_tecplot360_export_temperature(t.maxnod, t.pa, t.maxelm, t.nvtx, t.potent,t,0);
			}

			

		}

		// steady Static Structural
		if (1 && steady_or_unsteady_global_determinant == 5) {

			// ����� �������.
			unsigned int calculation_start_time = 0; // ������ ����� ��.
			unsigned int calculation_end_time = 0; // ��������� ����� ��.
			unsigned int calculation_seach_time = 0; // ����� ���������� ������� ���� � ��.

			calculation_start_time = clock(); // ������ ������ �����.

			for (integer i7 = 0; i7<t.maxelm + t.maxbound; i7++) t.potent[i7] = operating_temperature_for_film_coeff; // �������������.

		    // ������ ������ Static Structural.
			bonly_solid_calculation = true;

			// �������� ����������� ���������� �� ����������� ������.
			if (lw == 1) {
				bPhysics_stop = true;
				if (lb < 11) {
					// ��� ����������� �������� :
					// MD40, AuSn, Cu, AuSn, SiC, GaN. cabinet and hollow.
					bPhysics_PTBSH_memory = true;
				}
			}

			if (adiabatic_vs_heat_transfer_coeff == 1) {
				// �� �������������� ��������� ��������� ����� ��������� �� ��������������� ��� ������ ������ ����.
				//for (integer i7 = 0; i7<t.maxelm + t.maxbound; i7++) t.potent[i7] = 0.57*operating_temperature_for_film_coeff;
			}

			// ����� �������������� ��� �� ������ ������������ ������ ������ ����������������.
			bsolid_static_only = true;
			bool bcleantemp = false;
			if (eqin.itemper == 1) {
				bcleantemp = true;
				integer i = 0; // ������� �����
				for (i = 0; i < flow_interior; i++) {
					if (eqin.fluidinfo[i].iflow == 1) bcleantemp = false;
				}
				// ���� bcleantemp==true �� �� ������ ������ ������ ������������� ��� ����� ���������.
			}
			if (1 || bcleantemp) {
				// ������� ������������ ���������� (��� ��������) ������ ������ ���������������� � ��������� �������. 
				printf("solution of pure Static Structural...\n");
				printf("please, press any key to continue...\n");
				if (bwait) {
					//getchar();
					system("pause");
				}

				// ��� ������������ ������������� ����������� ��������.
				bool bprintmessage = true; // �������� �� ��������� �� �������.

				doublereal dbeta = 1.0; // ������ ������� ������������� �� �������.
				bool bmyconvective = false;
				if (starting_speed_Vx*starting_speed_Vx + starting_speed_Vy*starting_speed_Vy + starting_speed_Vz*starting_speed_Vz > 1.0e-30) {
					bmyconvective = true;
				}
				else {
					// �������� ������������� ��������� ��������.
					errno_t err_inicialization_data;
					FILE* fp_inicialization_data;
					err_inicialization_data = fopen_s(&fp_inicialization_data, "load.txt", "r");
					if (err_inicialization_data == 0) {
						// �������� ������ � ���� ������������.
						bmyconvective = true;
						fclose(fp_inicialization_data);
					}
				}

				// if (flow_interior>0) bmyconvective=true;
				// ������ ���������� ����������,
				// ��������� ��� �������� ������� ���-���
				doublereal **rhie_chow = NULL;
				QuickMemVorst m;
				m.ballocCRSt = false; // �������� ������
				m.bsignalfreeCRSt = true; // � ����� �����������.
										  // ������������� ����������.
				m.tval = NULL;
				m.tcol_ind = NULL;
				m.trow_ptr = NULL;
				m.tri = NULL;
				m.troc = NULL;
				m.ts = NULL;
				m.tt = NULL;
				m.tvi = NULL;
				m.tpi = NULL;
				m.tdx = NULL;
				m.tdax = NULL;
				m.ty = NULL;
				m.tz = NULL;
				m.ta = NULL;
				m.tja = NULL;
				m.tia = NULL;
				m.talu = NULL;
				m.tjlu = NULL;
				m.tju = NULL;
				m.tiw = NULL;
				m.tlevs = NULL;
				m.tw = NULL;
				m.tjw = NULL;
				m.icount_vel = 100000; // ����� ������� �����.

				bPhysics_stop = false;
				//bPhysics_stop = false;
				// ����� ������� Static Structural.
				solve_Structural(t, w, lw, m, false, operatingtemperature);
				//bPhysics_stop = true;

				/*
				// ���� flow_interior == 0 �� f[0] ������ ���������� ��������
				solve_nonlinear_temp(f[0], f, t,
				rhie_chow,
				b, lb, s, ls, w, lw,
				dbeta, flow_interior,
				bmyconvective, NULL, 0.001, 0.001,
				false,
				matlist, 0,
				bprintmessage,
				gtdps, ltdp, 1.0, m,
				NULL, // �������� � ����������� ���������� ����.
				NULL); // �������� ����� ����� ������� � ����������� ���������� ����.
				// ��������� �������� ������ 1.0 �������� ��� �������� �������.
				*/
				// ���������� ����� ������.
				massa_cabinet(t, f, inx, iny, inz,
					xpos, ypos, zpos, flow_interior,
					b, lb, operatingtemperature,
					matlist);


				calculation_end_time = clock(); // ������ ��������� �����.
				calculation_seach_time = calculation_end_time - calculation_start_time;
				unsigned int im = 0, is = 0, ims = 0;
				im = (unsigned int)(calculation_seach_time / 60000); // ������
				is = (unsigned int)((calculation_seach_time - 60000 * im) / 1000); // �������
				ims = (unsigned int)((calculation_seach_time - 60000 * im - 1000 * is) / 10); // ������������ ������� �� 10

				printf("time calculation is:  %d minute %d second %d millisecond\n", im, is, 10 * ims);

				if (1) {
					if (!b_on_adaptive_local_refinement_mesh) {
						// ������� ���������� ���������� � ��������� tecplot360:
						exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint, 0);
					}
					else {
						// ������� � ��������� ������� �����������.
						//� ���� �����.
						ANES_tecplot360_export_temperature(t.maxnod, t.pa, t.maxelm, t.nvtx, t.potent, t,0);
					}
				}

			}

			doublereal tmaxfinish = -273.15; // ���������� ����.
											 // ���������� �������� ������������ ����������� ������ ��������� ������� � �� � ��������:
											 //for (integer i = 0; i < t.maxelm + t.maxbound; i++) tmaxfinish = fmax(tmaxfinish, fabs(t.potent[i]));
											 // 23 ������� 2015
											 // �� ��������� ������ ���������� ����� �� ����� ��������� ������� �����������, �������
											 // �������� �� ������� ����� � ��������� ����������� ������ �� ���������� ��. 
			for (integer i = 0; i < t.maxelm; i++) tmaxfinish = fmax(tmaxfinish, t.potent[i]);

			doublereal totaldeform_max = -1.0e+30;
			for (integer i = 0; i < t.maxelm; i++) totaldeform_max = fmax(totaldeform_max, t.total_deformation[TOTALDEFORMATION][i]);

			FILE *fp;
			errno_t err1;
			err1 = fopen_s(&fp, "report.txt", "w");
			// �������� ����� ��� ������.
			if ((err1) != 0) {
				printf("Create File report.txt Error\n");
				//getchar();
				system("pause");
			}
			else {
				// ������ ���������
				fprintf(fp, "Maximum Temperature %.2f\n", tmaxfinish);
				fclose(fp);
			}
			// 1 - solver/solid_static/
			report_temperature(flow_interior, f, t, b, lb, s, ls, w, lw, 0);

			

		}

		// steady Static Structural and Temperature (Thermal Stress).
		if (1 && steady_or_unsteady_global_determinant == 6) {

			// ����� �������.
			unsigned int calculation_start_time = 0; // ������ ����� ��.
			unsigned int calculation_end_time = 0; // ��������� ����� ��.
			unsigned int calculation_seach_time = 0; // ����� ���������� ������� ���� � ��.

			calculation_start_time = clock(); // ������ ������ �����.

			for (integer i7 = 0; i7<t.maxelm + t.maxbound; i7++) t.potent[i7] = operating_temperature_for_film_coeff; // �������������.

			// ������ �������������, � ����� Static Structural.
			bonly_solid_calculation = true;

			// �������� ����������� ���������� �� ����������� ������.
			if (lw == 1) {
				bPhysics_stop = true;
				if (lb < 11) {
					// ��� ����������� �������� :
					// MD40, AuSn, Cu, AuSn, SiC, GaN. cabinet and hollow.
					bPhysics_PTBSH_memory = true;
				}
			}

			if (adiabatic_vs_heat_transfer_coeff == 1) {
				// �� �������������� ��������� ��������� ����� ��������� �� ��������������� ��� ������ ������ ����.
				//for (integer i7 = 0; i7<t.maxelm + t.maxbound; i7++) t.potent[i7] = 0.57*operating_temperature_for_film_coeff;
			}

			// ����� �������������� ��� �� ������ ������������ ������ ������ ����������������.
			bsolid_static_only = true;
			bool bcleantemp = false;
			if (eqin.itemper == 1) {
				bcleantemp = true;
				integer i = 0; // ������� �����
				for (i = 0; i < flow_interior; i++) {
					if (eqin.fluidinfo[i].iflow == 1) bcleantemp = false;
				}
				// ���� bcleantemp==true �� �� ������ ������ ������ ������������� ��� ����� ���������.
			}
			if (1 || bcleantemp) {
				// ������� ������������ ���������� (��� ��������) ������ ������ ���������������� � ��������� �������. 
				printf("solution of pure Static Structural...\n");
				printf("please, press any key to continue...\n");
				if (bwait) {
					//getchar();
					system("pause");
				}

				// ��� ������������ ������������� ����������� ��������.
				bool bprintmessage = true; // �������� �� ��������� �� �������.

				doublereal dbeta = 1.0; // ������ ������� ������������� �� �������.
				bool bmyconvective = false;
				if (starting_speed_Vx*starting_speed_Vx + starting_speed_Vy*starting_speed_Vy + starting_speed_Vz*starting_speed_Vz > 1.0e-30) {
					bmyconvective = true;
				}
				else {
					// �������� ������������� ��������� ��������.
					errno_t err_inicialization_data;
					FILE* fp_inicialization_data;
					err_inicialization_data = fopen_s(&fp_inicialization_data, "load.txt", "r");
					if (err_inicialization_data == 0) {
						// �������� ������ � ���� ������������.
						bmyconvective = true;
						fclose(fp_inicialization_data);
					}
				}

				// if (flow_interior>0) bmyconvective=true;
				// ������ ���������� ����������,
				// ��������� ��� �������� ������� ���-���
				doublereal **rhie_chow = NULL;
				QuickMemVorst m;
				m.ballocCRSt = false; // �������� ������
				m.bsignalfreeCRSt = true; // � ����� �����������.
										  // ������������� ����������.
				m.tval = NULL;
				m.tcol_ind = NULL;
				m.trow_ptr = NULL;
				m.tri = NULL;
				m.troc = NULL;
				m.ts = NULL;
				m.tt = NULL;
				m.tvi = NULL;
				m.tpi = NULL;
				m.tdx = NULL;
				m.tdax = NULL;
				m.ty = NULL;
				m.tz = NULL;
				m.ta = NULL;
				m.tja = NULL;
				m.tia = NULL;
				m.talu = NULL;
				m.tjlu = NULL;
				m.tju = NULL;
				m.tiw = NULL;
				m.tlevs = NULL;
				m.tw = NULL;
				m.tjw = NULL;
				m.icount_vel = 100000; // ����� ������� �����.

				


				
				// ���� flow_interior == 0 �� f[0] ������ ���������� ��������
				solve_nonlinear_temp(f[0], f, t,
				rhie_chow,
				b, lb, s, ls, w, lw,
				dbeta, flow_interior,
				bmyconvective, NULL, 0.001, 0.001,
				false,
				matlist, 0,
				bprintmessage,
				gtdps, ltdp, 1.0, m,
				NULL, // �������� � ����������� ���������� ����.
				NULL); // �������� ����� ����� ������� � ����������� ���������� ����.
				// ��������� �������� ������ 1.0 �������� ��� �������� �������.
				
				//bPhysics_stop = false;
				// ����� ������� Static Structural.
				solve_Structural(t, w, lw, m, true, operatingtemperature);
				//bPhysics_stop = true;

				// ���������� ����� ������.
				massa_cabinet(t, f, inx, iny, inz,
					xpos, ypos, zpos, flow_interior,
					b, lb, operatingtemperature,
					matlist);

				calculation_end_time = clock(); // ������ ��������� �����.
				calculation_seach_time = calculation_end_time - calculation_start_time;
				unsigned int im = 0, is = 0, ims = 0;
				im = (unsigned int)(calculation_seach_time / 60000); // ������
				is = (unsigned int)((calculation_seach_time - 60000 * im) / 1000); // �������
				ims = (unsigned int)((calculation_seach_time - 60000 * im - 1000 * is) / 10); // ������������ ������� �� 10

				printf("time calculation is:  %d minute %d second %d millisecond\n", im, is, 10 * ims);

				if (1) {
					if (!b_on_adaptive_local_refinement_mesh) {
						// ������� ���������� ���������� � ��������� tecplot360:
						exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint, 0);
					}
					else {
						// ������� � ��������� ������� �����������.
						//� ���� �����.
						ANES_tecplot360_export_temperature(t.maxnod, t.pa, t.maxelm, t.nvtx, t.potent, t,0);
					}
				}

			}

			doublereal tmaxfinish = -273.15; // ���������� ����.
											 // ���������� �������� ������������ ����������� ������ ��������� ������� � �� � ��������:
											 //for (integer i = 0; i < t.maxelm + t.maxbound; i++) tmaxfinish = fmax(tmaxfinish, fabs(t.potent[i]));
											 // 23 ������� 2015
											 // �� ��������� ������ ���������� ����� �� ����� ��������� ������� �����������, �������
											 // �������� �� ������� ����� � ��������� ����������� ������ �� ���������� ��. 
			for (integer i = 0; i < t.maxelm; i++) tmaxfinish = fmax(tmaxfinish, t.potent[i]);

			doublereal totaldeform_max = -1.0e+30;
			for (integer i = 0; i < t.maxelm; i++) totaldeform_max = fmax(totaldeform_max, t.total_deformation[TOTALDEFORMATION][i]);

			FILE *fp;
			errno_t err1;
			err1 = fopen_s(&fp, "report.txt", "w");
			// �������� ����� ��� ������.
			if ((err1) != 0) {
				printf("Create File report.txt Error\n");
				//getchar();
				system("pause");
			}
			else {
				// ������ ���������
				fprintf(fp, "Maximum Temperature %.2f\n", tmaxfinish);
				fclose(fp);
			}
			// 1 - solver/solid_static/
			report_temperature(flow_interior, f, t, b, lb, s, ls, w, lw, 0);

			

		}



		/*
		if (b_on_adaptive_local_refinement_mesh) {
		printf("t.maxbound=%d\n", t.maxbound);
		printf("v dvuch shagah ot ALICE sborki. \n");
		getchar();
		exit(1);  // ����� ����������� ������������ ��������� ���������� ���������� �������� ������������ �����.
		}

		if (b_on_adaptive_local_refinement_mesh) {
		printf("Solve temperature is compleate. \n");
		getchar();
		exit(1);  // ����� ����������� ������������ ��������� ���������� ���������� �������� ������������ �����.
		}
		*/

		//system("pause");

		if (1 && steady_or_unsteady_global_determinant == 1) {
			// �������������� ����������������.
			
			// ������ ������ ������������� � ������ ����.
			bonly_solid_calculation = true;

			// �������� ����������� ���������� �� ����������� ������.
			// ��� ���������� �������� ������ �� ���� �������� �������� ����������.
			if (lw == 1) {
				bPhysics_stop = true;
				if (lb < 11) {
					// ��� ����������� �������� :
					// MD40, AuSn, Cu, AuSn, SiC, GaN. cabinet and hollow.
					bPhysics_PTBSH_memory = true;
				}
			}

			bglobal_unsteady_temperature_determinant = true;
			// ����� �������������� ��� �� ������ ������ ������ ����������������.
			// ��������� 13 �������� bconvective
			// �� ����������� �������� ��������, �������� ������� ����������� ����������,
			// ����������� �� ����������� ��������� �����. ������� ������� ������������� �� 
			// ������� ������� ����������� ������ �����.
			// ��� ������������� ����� � ������������� ��������� 
			// ���������� ��������� ������� ������������� � ������� �� ������
			// ��� ��������� ����������� � ������ ������� �� 5 �������� �� ���� ��������� � 167�������� � �������.
			// ��� ��������� �������� � icepak ����� 120 ��������, ��� ������ � ��������� ����������� Rt 
			// (�������� 6.875��, RT=16K/W) � �����������������
			// ������.
			doublereal dbeta = 1.3333333;//1.0; // ���� 1.0 �� ������ ������� ������������� �� �������.
			dbeta = 1.0; // ����� ���������� ��������.
			// ������ ���������� ����������,
			// ��������� ��� �������� ������� ���-���
			doublereal **rhie_chow = NULL;
			//solve_nonlinear_temp(f[0], f, t, rhie_chow, b, lb, s, ls, w, lw, dbeta, flow_interior, false, NULL, 0.001, false);
			unsteady_temperature_calculation(f[0], f, t,
				rhie_chow,
				b, lb, s, ls, w, lw,
				dbeta, flow_interior,
				false, matlist,
				operatingtemperature,
				gtdps, ltdp); // �������������� ������������� ������

			// ���������� ����� ������.
			massa_cabinet(t, f, inx, iny, inz,
				xpos, ypos, zpos, flow_interior,
				b, lb, operatingtemperature,
				matlist);

			if (!b_on_adaptive_local_refinement_mesh) {
				// ������� ���������� ���������� � ��������� tecplot360:
				exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint,0);
			}
			else {
				// ������� � ��������� ������� �����������.
				//� ���� �����.
				ANES_tecplot360_export_temperature(t.maxnod, t.pa, t.maxelm, t.nvtx, t.potent,t,0);
			}

			doublereal tmaxfinish = -273.15;
			// ���������� �������� ������������ ����������� ������ ��������� ������� � �� � ��������:
			for (integer i = 0; i < t.maxelm + t.maxbound; i++) tmaxfinish = fmax(tmaxfinish, t.potent[i]);
			FILE *fp;
			errno_t err1;
			err1 = fopen_s(&fp, "report.txt", "w");
			// �������� ����� ��� ������.
			if ((err1) != 0) {
				printf("Create File report.txt Error\n");
				//getchar();
				system("pause");
			}
			else {
				// ������ ���������
				fprintf(fp, "Maximum Temperature %.2f\n", tmaxfinish);
				fclose(fp);
			}
			// 1 - solver/solid_static/
			report_temperature(flow_interior, f, t, b, lb, s, ls, w, lw, 0);

			printf("calculation complete...\n");
			// getchar();
		}

		fclose(fp_radiation_log);

		// ������� ���������� ���������� � ��������� tecplot360:
		// ����� ������������ ��� �������� ����������� �����.
		if (false) {
			exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint,0);
			printf("read values. OK.\n");
			if (bwait) {
				//getchar(); // debug avtosave
				system("pause");
			}
		}

		if ((1 && steady_or_unsteady_global_determinant == 3)) {
			// Fluid dynamic.

			told_temperature_global_for_HOrelax = new doublereal[t.maxelm + t.maxbound];
			bSIMPLErun_now_for_temperature = true;
			if (dgx*dgx + dgy*dgy + dgz*dgz > 1.0e-20) {
				// ���� ����� ��������� �������� �� ��� fluid ���������� ����������� ����������.
				bool bbussinesk_7 = false;
#pragma omp parallel for
				for (integer i_8 = 0; i_8 < f[0].maxelm; i_8++) {
					integer ib = t.whot_is_block[f[0].ptr[i_8]];
					if (ib > -1) {
						if (b[ib].itype == FLUID) {
							integer i_7 = b[ib].imatid;
							if (matlist[i_7].bBussineskApproach) bbussinesk_7 = true;
						}
					}
				}
				if (bbussinesk_7) {					
					bSIMPLErun_now_for_natural_convection = true;
				}
			}
			bHORF = true;
			bPamendment_source_old = new doublereal[f[0].maxelm + f[0].maxbound];
			for (integer i5 = 0; i5 < f[0].maxelm + f[0].maxbound; i5++) bPamendment_source_old[i5] = 0.0;
			// exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint);
			//getchar();
			if (dgx*dgx + dgy*dgy + dgz*dgz > 1.0e-20) {
				// ���� ����� ��������� �������� �� ��� fluid ���������� ����������� ����������.
				bool bbussinesk_7 = false;
#pragma omp parallel for
				for (integer i_8 = 0; i_8 < f[0].maxelm; i_8++) {
					integer ib = t.whot_is_block[f[0].ptr[i_8]];
					if (ib > -1) {
						if (b[ib].itype == FLUID) {
							integer i_7 = b[ib].imatid;
							if (matlist[i_7].bBussineskApproach) {
								bbussinesk_7 = true;
							}
						}
					}
				}
				if (bbussinesk_7) {
					printf("Bussinesk approach Operating Temperature=%e\n", f[0].OpTemp); // Operating Temperature);
				}
			}

			//exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint, 0);
			//getchar();

			// ������������ ����������������� ��������.
			steady_cfd_calculation(breadOk,
				eqin, dgx, dgy, dgz,
				continity_start,
				inumber_iteration_SIMPLE,
				flow_interior, f, t, b, lb,
				s, ls, w, lw, matlist,
				gtdps, ltdp, bextendedprint);
			//xyplot( f, 0, t);
			// boundarylayer_info(f, t, flow_interior, w, lw);
			// 2 - solver/conjugate_heat_transfer_static/
			report_temperature(flow_interior, f, t, b, lb, s, ls, w, lw, 0/*2*/);

			// ���������� ����� ������.
			massa_cabinet(t, f, inx, iny, inz,
				xpos, ypos, zpos, flow_interior,
				b, lb, operatingtemperature,
				matlist);

			// ������� ���������� ���������� � ��������� tecplot360:
			exporttecplotxy360T_3D_part2(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint,0);
			save_velocity_for_init(t.maxelm, t.ncell, f, t, flow_interior);
			// exporttecplotxy360T_3D_part2_rev(t.maxelm, t.ncell, f, t, flow_interior, 0, bextendedprint,b,lb);
			delete[] bPamendment_source_old;
			delete[] told_temperature_global_for_HOrelax;
		}
		if (0) {
			

			told_temperature_global_for_HOrelax = new doublereal[t.maxelm + t.maxbound];
			bSIMPLErun_now_for_temperature = true;
			bHORF = true;
			bPamendment_source_old = new doublereal[f[0].maxelm + f[0].maxbound];
			for (integer i5 = 0; i5 < f[0].maxelm + f[0].maxbound; i5++) bPamendment_source_old[i5] = 0.0;
			// �������������� ����������������� �������� :
			usteady_cfd_calculation(breadOk, eqin,
				dgx, dgy, dgz,
				continity_start,
				inumber_iteration_SIMPLE,
				flow_interior,
				f, t,
				b, lb, s, ls,
				w, lw, matlist, gtdps, ltdp, bextendedprint);
			delete[] bPamendment_source_old;
			delete[] told_temperature_global_for_HOrelax;

			// ���������� ����� ������.
			massa_cabinet(t, f, inx, iny, inz,
				xpos, ypos, zpos, flow_interior,
				b, lb, operatingtemperature,
				matlist);
		}

		fclose(fp_log); // �������� ����� ����.


		if (continity_start != NULL) {
			delete[] continity_start;
			continity_start = NULL;
		}

		if (inumber_iteration_SIMPLE != NULL) {
			delete[] inumber_iteration_SIMPLE;
			inumber_iteration_SIMPLE = NULL;
		}

	}

	// ������������ ����������� ������.
	if (t.xpos_copy != NULL) {
		delete[] t.xpos_copy;
		t.xpos_copy = NULL;
	}
	if (t.ypos_copy != NULL) {
		delete[] t.ypos_copy;
		t.ypos_copy = NULL;
	}
	if (t.zpos_copy != NULL) {
		delete[] t.zpos_copy;
		t.zpos_copy = NULL;
	}
	
	// ������������ ����������� ������.
	if (xposadd != NULL) {
		delete[] xposadd;
		xposadd = NULL;
	}
	if (yposadd != NULL) {
		delete[] yposadd;
		yposadd = NULL;
	}
	if (zposadd != NULL) {
		delete[] zposadd;
		zposadd = NULL;
	}

	
	// ������������ ����������� ������.
	if (xpos != NULL) {
		delete[] xpos;
		xpos = NULL;
	}
	if (ypos != NULL) {
		delete[] ypos;
		ypos = NULL;
	}
	if (zpos != NULL) {
		delete[] zpos;
		zpos = NULL;
	}

	if (bsource_term_radiation_for_relax != NULL) {
		delete[] bsource_term_radiation_for_relax; // ���������� ������������ ������ ������������ �������.
		bsource_term_radiation_for_relax = NULL;
	}
	if (b_buffer_correct_source != NULL) {
		delete[] b_buffer_correct_source;
		b_buffer_correct_source = NULL;
	}

	printf("free memory begin...\n");
	if (bwait) {
		//getchar();
		system("pause");
	}

	if (rthdsd_no_radiosity_patch != NULL) {
		free(rthdsd_no_radiosity_patch);
	}
	rthdsd_no_radiosity_patch = NULL;

	// ������� ��������� ���������� ��������� ������� � ����� 0.14 ��������.
	if (qnbc != NULL) {
		delete[] qnbc;
		qnbc = NULL;
		iadd_qnbc_maxelm = 0;
	}

	// ������������ ������ �� ��� amg1r5.
	if (amgGM.a != NULL) {
		delete amgGM.a;
		amgGM.a = NULL;
	}
	if (amgGM.ia != NULL) {
		delete amgGM.ia;
		amgGM.ia = NULL;
	}
	if (amgGM.ja != NULL) {
		delete amgGM.ja;
		amgGM.ja = NULL;
	}
	if (amgGM.u != NULL) {
		delete amgGM.u;
		amgGM.u = NULL;
	}
	if (amgGM.f != NULL) {
		delete amgGM.f;
		amgGM.f = NULL;
	}
	if (amgGM.ig != NULL) {
		delete amgGM.ig;
		amgGM.ig = NULL;
	}

	amgGM.nda = -1;
	amgGM.ndf = -1;
	amgGM.ndia = -1;
	amgGM.ndig = -1;
	amgGM.ndja = -1;
	amgGM.ndu = -1;

	for (integer i_7 = 0; i_7 < lb; i_7++) {
		if (b[i_7].temp_Sc != NULL) {
			delete[] b[i_7].temp_Sc;
			b[i_7].temp_Sc = NULL;
		}
		if (b[i_7].arr_Sc != NULL) {
			delete[] b[i_7].arr_Sc;
			b[i_7].arr_Sc = NULL;
		}
		if (b[i_7].g.hi != NULL) {
			delete[] b[i_7].g.hi;
			b[i_7].g.hi = NULL;
		}
		if (b[i_7].g.xi != NULL) {
			delete[] b[i_7].g.xi;
			b[i_7].g.xi = NULL;
		}
		if (b[i_7].g.yi != NULL) {
			delete[] b[i_7].g.yi;
			b[i_7].g.yi = NULL;
		}
		if (b[i_7].g.zi != NULL) {
			delete[] b[i_7].g.zi;
			b[i_7].g.zi = NULL;
		}
	}
	delete b; delete s; delete w; // ������������ ������
	for (integer i_7 = 0; i_7 < lmatmax; i_7++) {
		if (matlist[i_7].arr_cp != NULL) {
			delete[] matlist[i_7].arr_cp;
			matlist[i_7].arr_cp = NULL;
     	}
		if (matlist[i_7].temp_cp != NULL) {
			delete[] matlist[i_7].temp_cp;
			matlist[i_7].temp_cp = NULL;
		}
		if (matlist[i_7].arr_lam != NULL) {
			delete[] matlist[i_7].arr_lam;
			matlist[i_7].arr_lam = NULL;
		}
		if (matlist[i_7].temp_lam != NULL) {
			delete[] matlist[i_7].temp_lam;
			matlist[i_7].temp_lam = NULL;
		}
	}
	delete[] matlist;
	delete[] gtdps;
	if (eqin.fluidinfo != NULL) {
		delete eqin.fluidinfo;
		eqin.fluidinfo = NULL;
	}
	// ����� ���������� ����������� ������ �� ��� ���� �������� ������:
	free_level1_temp(t);
	free_level2_temp(t); // ������������ ������ �� ��� ������.
	// ����������� ������ ��� LR ������.
	free_root(t.rootWE, t.iWE);
	free_root(t.rootSN, t.iSN);
	free_root(t.rootBT, t.iBT);
	if (t.rootWE != NULL) {
		delete[] t.rootWE;
		t.rootWE = NULL;
	}
	if (t.rootSN != NULL) {
		delete[] t.rootSN;
		t.rootSN = NULL;
	}
	if (t.rootBT != NULL) {
		delete[] t.rootBT;
		t.rootBT = NULL;
	}
	// ������������ ������ ��� LR �����.
	free_level1_flow(f, flow_interior);
	free_level2_flow(f, flow_interior); // ������������ ������ �� ��� ������.

	delete[] f;
	f = NULL;

	if (sourse2Dproblem != NULL) {
		delete[] sourse2Dproblem;
		sourse2Dproblem = NULL;
	}
	if (conductivity2Dinsource != NULL) {
		delete[] conductivity2Dinsource;
		conductivity2Dinsource = NULL;
	}

	if (x_jacoby_buffer != NULL) {
		// 30 ������� 2016. 
		// � seidelsor2 ������ ������������� �� ����� ������ ���������� �.�. �����.
		// ������������ ������ �� ��� jacobi buffer.
		delete[] x_jacoby_buffer;
	}

	if (bvery_big_memory) {
		if (database.x != NULL) {
			free(database.x);
		}
		if (database.y != NULL) {
			free(database.y);
		}
		if (database.z != NULL) {
			free(database.z);
		}
		if (database.nvtxcell != NULL) {
			for (integer i = 0; i <= 7; i++) {
				delete[] database.nvtxcell[i];
			}
			delete[] database.nvtxcell;
		}
		if (database.ptr != NULL) {
			if (database.ptr[0] != NULL) {
				delete[] database.ptr[0];
			}
			if (database.ptr[1] != NULL) {
				delete[] database.ptr[1];
			}
			delete[] database.ptr;
		}
	}

	// ������������ ����� ������ � ILU �������.
	if (milu_gl_buffer.alu_copy != NULL) delete[] milu_gl_buffer.alu_copy;
	if (milu_gl_buffer.jlu_copy != NULL) delete[] milu_gl_buffer.jlu_copy;
	if (milu_gl_buffer.ju_copy != NULL) delete[] milu_gl_buffer.ju_copy;
	milu_gl_buffer.alu_copy = NULL;
	milu_gl_buffer.jlu_copy = NULL;
	milu_gl_buffer.ju_copy = NULL;



	flow_interior = 0;
	printf("free memory finish...\n");

	if (1 && steady_or_unsteady_global_determinant == 2) {
		// ��� ������ ����� ��������� ����������.
		printf("Mesh generation procedure is finish.\n");
	}
	else {
		printf("Calculation procedure is finish.\n");
	}
	printf("Please, press any key to exit...\n");
	if (bwait) {
		//getchar();
		system("pause");
	}

	

	calculation_main_end_time = clock();
	calculation_main_seach_time = calculation_main_end_time - calculation_main_start_time;


	/*printf("time=%d statistic vorst=%3.2f %% \n",calculation_main_seach_time,(float)(100.0*calculation_vorst_seach_time/calculation_main_seach_time));
	getchar();
	*/

	
	
	// ����� ����� ����������.
	int im=0, is=0, ims=0;
	im=(int)(calculation_main_seach_time/60000); // ������
	is=(int)((calculation_main_seach_time-60000*im)/1000); // �������
	ims=(int)((calculation_main_seach_time-60000*im-1000*is)/10); // ������������ ������� �� 10
	
	printf("time calculation is:  %d minute %d second %d millisecond\n", im, is, 10*ims);


	system("pause");
	return 0;
}