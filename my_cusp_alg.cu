
#include <hip/hip_runtime.h>

//CUSP 0.5.1 library
// my_cusp_alg.cu

#define GPU_LIB_INCLUDE_MY_PROJECT 0

// amgx
//#include "amgx_c.h"

/*
#include <cusp/csr_matrix.h>
#include <cusp/krylov/bicgstab.h>

#if defined(__cplusplus)
extern "C" {
#endif

void bicgstab_(integer * device_I, integer * device_J, float * device_V, float * device_x, float * device_b, integer N, integer NNZ){

// *NOTE* raw pointers must be wrapped with thrust::device_ptr!
thrust::device_ptr<int> wrapped_device_I(device_I);
thrust::device_ptr<int> wrapped_device_J(device_J);
thrust::device_ptr<float> wrapped_device_V(device_V);
thrust::device_ptr<float> wrapped_device_x(device_x);
thrust::device_ptr<float> wrapped_device_b(device_b);

// use array1d_view to wrap the individual arrays
typedef typename cusp::array1d_view< thrust::device_ptr<int> > DeviceIndexArrayView;
typedef typename cusp::array1d_view< thrust::device_ptr<float> > DeviceValueArrayView;

DeviceIndexArrayView row_indices (wrapped_device_I, wrapped_device_I + (N+1));
DeviceIndexArrayView column_indices(wrapped_device_J, wrapped_device_J + NNZ);
DeviceValueArrayView values (wrapped_device_V, wrapped_device_V + NNZ);
DeviceValueArrayView x (wrapped_device_x, wrapped_device_x + N);
DeviceValueArrayView b (wrapped_device_b, wrapped_device_b + N);

// combine the three array1d_views into a csr_matrix_view
typedef cusp::csr_matrix_view<DeviceIndexArrayView,
DeviceIndexArrayView,
DeviceValueArrayView> DeviceView;

// construct a csr_matrix_view from the array1d_views
DeviceView A(N, N, NNZ, row_indices, column_indices, values);

// set stopping criteria:
// iteration_limit = 100
// relative_tolerance = 1e-5
cusp::verbose_monitor<float> monitor(b, 100, 1e-5);

// solve the linear system A * x = b with the Conjugate Gradient method
cusp::krylov::bicgstab(A, x, b, monitor);

}

#if defined(__cplusplus)
}
#endif

*/

#if GPU_LIB_INCLUDE_MY_PROJECT == 1 

// CUSP 0.5.1 NVIDIA Includes

//#include "cusp_library\cusp\csr_matrix.h"
//#include "cusp_library\cusp/krylov/bicgstab.h"
//#include  <cusp\csr_matrix.h>
//#include  <cusp/krylov/bicgstab.h>
//#include <cusp/hyb_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/monitor.h>
#include <cusp/array1d.h>
//#include <cusp/io/matrix_market.h>
//#include <cusp/krylov/cg.h>
#include <cusp/precond/aggregation/smoothed_aggregation.h>
#include <cusp/precond/ainv.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/krylov/gmres.h>
#include <cusp/precond/aggregation/smoothed_aggregation_options.h>
#include <cusp/csr_matrix.h>
#include <cusp/blas/blas.h>
#include <cusp/linear_operator.h>
//#include <cusp/gallery/poisson.h>
#include <cusp/relaxation/gauss_seidel.h>

#endif


#if GPU_LIB_INCLUDE_MY_PROJECT==1

//bool bgl_first_start_nonlinear_cusp_amg = true;

bool bstart7 = true;

#if 1
// ��� ����� ������������� �������� ������ �������� �������������� ���������.
// ���������� Cusp ������ 0.5.1. ��� ���������� � �������� �������� ����� ������������������ 
// �� open Source �������� Apache license 2.0. 
// �� ����� (����������� ����������) ��������� ����� bicgstab � �������������� ������������� �����
// �� ������ ���������� ���������� samg.
// ���� ������������� 12 ������� 2016 ����.
void cusp_solver_amghost(equation3D* &sl, equation3D_bon* &slb,
	integer maxelm, integer maxbound,
	doublereal *dV, doublereal* &dX0, integer maxit,
	doublereal alpharelax, integer iVar)
{

	// maxit,  iVar - �� ������������.


	if (dX0 == NULL) {
		dX0 = new doublereal[maxelm + maxbound];
		for (integer i = 0; i < maxelm + maxbound; i++) {
			dX0[i] = 0.0;
		}
	}

	// TODO �������� val, col_ind, row_ptr
	integer nna = 0; // ���������� ��������� ��������� � ������� ����.

	const doublereal nonzeroEPS = 1e-37; // ��� ��������� ������������� ����

	// ������� ����� ��������� ��������� � �������.
	nna = 0;
	for (integer i = 0; i<maxelm; i++) {
		// ������������ �������.
		if ((sl[i].iP>-1) && (fabs(sl[i].ap) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB>-1) && (fabs(sl[i].ab) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE>-1) && (fabs(sl[i].ae) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN>-1) && (fabs(sl[i].an) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS>-1) && (fabs(sl[i].as) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT>-1) && (fabs(sl[i].at) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW>-1) && (fabs(sl[i].aw) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB2>-1) && (fabs(sl[i].ab2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE2>-1) && (fabs(sl[i].ae2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN2>-1) && (fabs(sl[i].an2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS2>-1) && (fabs(sl[i].as2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT2>-1) && (fabs(sl[i].at2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW2>-1) && (fabs(sl[i].aw2) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB3>-1) && (fabs(sl[i].ab3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE3>-1) && (fabs(sl[i].ae3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN3>-1) && (fabs(sl[i].an3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS3>-1) && (fabs(sl[i].as3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT3>-1) && (fabs(sl[i].at3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW3>-1) && (fabs(sl[i].aw3) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB4>-1) && (fabs(sl[i].ab4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE4>-1) && (fabs(sl[i].ae4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN4>-1) && (fabs(sl[i].an4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS4>-1) && (fabs(sl[i].as4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT4>-1) && (fabs(sl[i].at4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW4>-1) && (fabs(sl[i].aw4) > nonzeroEPS)) (nna)++;

	}
	for (integer i = 0; i<maxbound; i++) {
		// ��������� ����.
		if ((slb[i].iW>-1) && (fabs(slb[i].aw) > nonzeroEPS)) (nna)++;
		if ((slb[i].iI>-1) && (fabs(slb[i].ai) > nonzeroEPS)) (nna)++;
	}

	integer nnu = 0; // ����� �����������.
	nnu = maxelm + maxbound;

	typedef doublereal    ScalarType;  // feel free to change this to double if supported by your device
	//typedef float    ScalarType;
	typedef cusp::device_memory MemorySpace;


	// allocate storage for (nnu,nnu) matrix with nna nonzeros
	cusp::coo_matrix<int, ScalarType, cusp::host_memory> Ah(nnu, nnu, nna);
	//cusp::coo_matrix<int, float, cusp::device_memory> Ah(nnu, nnu, nna);

	//printf("0\n");
	//getchar();

	// initialize matrix entries on host
	nna = 0;
	//Ah.row_indices[0] = 0; Ah.column_indices[0] = 0; Ah.values[0] = 10.0; // demo interface
	for (integer i = 0; i<maxelm; i++) {
		// ������������ �������.
		if ((sl[i].iP>-1) && (fabs(sl[i].ap) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iP; Ah.values[nna] = sl[i].ap / alpharelax;
			(nna)++;
		}
		if ((sl[i].iB > -1) && (fabs(sl[i].ab) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB; Ah.values[nna] = -sl[i].ab;
			(nna)++;
		}
		if ((sl[i].iE > -1) && (fabs(sl[i].ae) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE; Ah.values[nna] = -sl[i].ae;
			(nna)++;
		}
		if ((sl[i].iN > -1) && (fabs(sl[i].an) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN; Ah.values[nna] = -sl[i].an;
			(nna)++;
		}
		if ((sl[i].iS > -1) && (fabs(sl[i].as) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS; Ah.values[nna] = -sl[i].as;
			(nna)++;
		}
		if ((sl[i].iT > -1) && (fabs(sl[i].at) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT; Ah.values[nna] = -sl[i].at;
			(nna)++;
		}
		if ((sl[i].iW > -1) && (fabs(sl[i].aw) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW; Ah.values[nna] = -sl[i].aw;
			(nna)++;
		}

		if ((sl[i].iB2 > -1) && (fabs(sl[i].ab2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB2; Ah.values[nna] = -sl[i].ab2;
			(nna)++;
		}
		if ((sl[i].iE2 > -1) && (fabs(sl[i].ae2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE2; Ah.values[nna] = -sl[i].ae2;
			(nna)++;
		}
		if ((sl[i].iN2 > -1) && (fabs(sl[i].an2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN2; Ah.values[nna] = -sl[i].an2;
			(nna)++;
		}
		if ((sl[i].iS2 > -1) && (fabs(sl[i].as2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS2; Ah.values[nna] = -sl[i].as2;
			(nna)++;
		}
		if ((sl[i].iT2 > -1) && (fabs(sl[i].at2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT2; Ah.values[nna] = -sl[i].at2;
			(nna)++;
		}
		if ((sl[i].iW2 > -1) && (fabs(sl[i].aw2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW2; Ah.values[nna] = -sl[i].aw2;
			(nna)++;
		}

		if ((sl[i].iB3 > -1) && (fabs(sl[i].ab3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB3; Ah.values[nna] = -sl[i].ab3;
			(nna)++;
		}
		if ((sl[i].iE3 > -1) && (fabs(sl[i].ae3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE3; Ah.values[nna] = -sl[i].ae3;
			(nna)++;
		}
		if ((sl[i].iN3 > -1) && (fabs(sl[i].an3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN3; Ah.values[nna] = -sl[i].an3;
			(nna)++;
		}
		if ((sl[i].iS3 > -1) && (fabs(sl[i].as3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS3; Ah.values[nna] = -sl[i].as3;
			(nna)++;
		}
		if ((sl[i].iT3 > -1) && (fabs(sl[i].at3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT3; Ah.values[nna] = -sl[i].at3;
			(nna)++;
		}
		if ((sl[i].iW3 > -1) && (fabs(sl[i].aw3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW3; Ah.values[nna] = -sl[i].aw3;
			(nna)++;
		}

		if ((sl[i].iB4 > -1) && (fabs(sl[i].ab4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB4; Ah.values[nna] = -sl[i].ab4;
			(nna)++;
		}
		if ((sl[i].iE4 > -1) && (fabs(sl[i].ae4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE4; Ah.values[nna] = -sl[i].ae4;
			(nna)++;
		}
		if ((sl[i].iN4 > -1) && (fabs(sl[i].an4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN4; Ah.values[nna] = -sl[i].an4;
			(nna)++;
		}
		if ((sl[i].iS4 > -1) && (fabs(sl[i].as4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS4; Ah.values[nna] = -sl[i].as4;
			(nna)++;
		}
		if ((sl[i].iT4 > -1) && (fabs(sl[i].at4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT4; Ah.values[nna] = -sl[i].at4;
			(nna)++;
		}
		if ((sl[i].iW4 > -1) && (fabs(sl[i].aw4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW4; Ah.values[nna] = -sl[i].aw4;
			(nna)++;
		}

	}

	for (integer i = 0; i<maxbound; i++) {
		// ��������� ����.
		if ((slb[i].iW>-1) && (fabs(slb[i].aw) > nonzeroEPS)) {
			Ah.row_indices[nna] = slb[i].iW; Ah.column_indices[nna] = slb[i].iW; Ah.values[nna] = slb[i].aw;
			(nna)++;
		}
		if ((slb[i].iI > -1) && (fabs(slb[i].ai) > nonzeroEPS)) {
			Ah.row_indices[nna] = slb[i].iW; Ah.column_indices[nna] = slb[i].iI; Ah.values[nna] = -slb[i].ai;
			(nna)++;
		}
	}


	cusp::array1d<ScalarType, cusp::host_memory> xh(nnu);
	cusp::array1d<ScalarType, cusp::host_memory> yh(nnu);
	//cusp::array1d<float, cusp::device_memory> xh(nnu);
	//cusp::array1d<float, cusp::device_memory> yh(nnu);

	//printf("0i5\n");
	//getchar();

	// yh = Ah x xh;
	for (integer i = 0; i < maxelm + maxbound; i++) {
		// b == dV[i];
		yh[i] = dV[i];
		xh[i] = 0.0;
		//xh[i] = dX0[i];
	}

	//printf("1\n");
	//getchar();
	// convert host->device
	//cusp::array1d<float, cusp::device_memory> xd = xh;
	//cusp::array1d<float, cusp::device_memory> xd(xh);
	//printf("2\n");
	//getchar();
	//cusp::array1d<float, cusp::device_memory> yd = yh;
	//cusp::array1d<float, cusp::device_memory> yd(yh);
	//printf("3\n");
	//getchar();

	//cusp::coo_matrix<int, float, cusp::device_memory> Ad = Ah;
	//cusp::coo_matrix<int, float, cusp::device_memory> Ad(Ah);
	//printf("4\n");
	//getchar();

	// ���������� ��������� 25 ������� 2016.
	doublereal tolerance = 1e-8;
	if (bSIMPLErun_now_for_temperature) {
		// ��� �������� ������� ��� CFD ����� ���� 
		// ������� ���������� �� ������ ��������� �������
		// ��������� ������ (���������� ������� cfd + temperature 
		// + ����������� ��������-����������.).
		switch (iVar) {
		case VX: tolerance = 1e-5;  break; //5e-5
		case VY: tolerance = 1e-5;  break; // 5e-5
		case VZ: tolerance = 1e-5;  break; // 5e-5
		case TEMP: tolerance = 1e-8;  break; // 1e-7
		case PAM: tolerance = 1e-7; break; // 1e-6
		}
	}

	integer imaxiter = 6000;
	if (((adiabatic_vs_heat_transfer_coeff > 0) || (breakRUMBAcalc_for_nonlinear_boundary_condition))) {
		// ��� ���������� �����. ������.
		// 348878024515312.94
		// 1947852997768.51
		imaxiter = 1;
		tolerance = 0.1;
	}

	if (bonly_solid_calculation) {
		tolerance = 1e-6;
		imaxiter = 6000;
	}

	/*
	if ((bonly_solid_calculation)&&(breakRUMBAcalc_for_nonlinear_boundary_condition)) {
	// 1 V - cycle,
	// ������������� �������� ������� 0.1.
	imaxiter = 1;
	tolerance = 0.1;
	getchar();
	}
	*/

	if (bstart7 && (((adiabatic_vs_heat_transfer_coeff > 0) || (breakRUMBAcalc_for_nonlinear_boundary_condition))))
	{
		imaxiter = 1;
		tolerance = 0.1;
		//cusp::relaxation::gauss_seidel(Ah, xh, yh, monitor); // 15_01_2017

		cusp::array1d<ScalarType, cusp::host_memory> rh(nnu);
		cusp::csr_matrix<int, ScalarType, cusp::host_memory> Ah7(Ah);

		// Construct gauss_seidel relaxation class
		cusp::relaxation::gauss_seidel<ScalarType, cusp::host_memory> Mh7(Ah7);
		// Compute initial residual
		cusp::multiply(Ah7, xh, rh);
		cusp::blas::axpy(yh, rh, ScalarType(-1));
		// Construct monitor with stopping criteria of 100 iterations or 1e-4 residual error
		cusp::monitor<ScalarType> monitor(yh, 100, 1e-4, 0, true);
		// Iteratively solve system
		while (!monitor.finished(rh))
		{
			Mh7(Ah7, yh, xh);
			cusp::multiply(Ah7, xh, rh);
			cusp::blas::axpy(yh, rh, ScalarType(-1));
			++monitor;
		}

		//bstart7 = false;

		if (bonly_solid_calculation) {
			// report status
			monitor.print();
		}
	}
	else {

		// set stopping criteria
		// iteration_limit = 100
		// relative_tolerance = 1e-6
		//  absolute_tolerance = 0
		//  verbose            = true
		//cusp::monitor<float> monitor(yd, 2000, 1e-6, 0, true);

		//cusp::default_monitor<float> monitor(yd, 6000, 1e-12);
		//-->cusp::default_monitor<ScalarType> monitor(yh, 6000, 1e-8);
		cusp::default_monitor<ScalarType> monitor(yh, imaxiter, tolerance);
		//cusp::default_monitor<float> monitor(yd, 2000, 1e-6);

		// setup preconditioner
		//cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::device_memory> Md(Ad);
		// setup preconditioner
		cusp::precond::aggregation::smoothed_aggregation<int, ScalarType, cusp::host_memory> Mh(Ah);
		//cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::device_memory> Md(Mh);
		// ������������ �������������������.
		//cusp::precond::diagonal<float, cusp::device_memory> Md(Ad);
		// AINV (NS Bridson).
		//cusp::precond::scaled_bridson_ainv<float, cusp::device_memory> Md(Ad);

		if (bonly_solid_calculation) {
			Mh.print();
		}

		// solve A * x = y to default tolerance with BiCGStab
		// with preconditioned BiCGStab
		//--->//cusp::krylov::bicgstab(Ad, xd, yd, monitor, Md);
		//cusp::krylov::bicgstab(Ad, xd, yd, monitor);
		cusp::krylov::bicgstab(Ah, xh, yh, monitor, Mh); // 15_01_2017
		//integer imy_restart_gmres = 20; // 20 � 50, 2000, 4000 �� ��������.
		//cusp::krylov::gmres(Ah, xh, yh, imy_restart_gmres, monitor, Mh); // �� ��������.

		if (bonly_solid_calculation) {
			// report status
			monitor.print();
		}

	}

	//cusp::array1d<float, cusp::host_memory> xh_ret = xd;
	//cusp::array1d<float, cusp::host_memory> xh_ret(xd);
	//printf("5\n");
	//getchar();



	// ����������� ����������.
	for (integer i = 0; i < maxelm + maxbound; i++) {
		//dX0[i]=xh_ret[i];
		dX0[i] = xh[i];
	}

} // cusp_solver_amghost

#endif


// ��� ����� ������������� �������� ������ �������� �������������� ���������.
// ���������� Cusp ������ 0.5.1. ��� ���������� � �������� �������� ����� ������������������ 
// �� open Source �������� Apache license 2.0. 
// �� ����� (����������� ����������) ��������� ����� bicgstab � �������������� ������������� �����
// �� ������ ���������� ���������� samg.
// ���� ������������� 12 ������� 2016 ����.
void cusp_solver_amgdevice(equation3D* &sl, equation3D_bon* &slb,
	integer maxelm, integer maxbound,
	doublereal *dV, doublereal* &dX0, integer maxit,
	doublereal alpharelax, integer iVar)
{

	// maxit,  iVar - �� ������������.


	if (dX0 == NULL) {
		dX0 = new doublereal[maxelm + maxbound];
		for (integer i = 0; i < maxelm + maxbound; i++) {
			dX0[i] = 0.0;
		}
	}

	// TODO �������� val, col_ind, row_ptr
	integer nna = 0; // ���������� ��������� ��������� � ������� ����.

	const doublereal nonzeroEPS = 1e-37; // ��� ��������� ������������� ����

	// ������� ����� ��������� ��������� � �������.
	nna = 0;
	for (integer i = 0; i<maxelm; i++) {
		// ������������ �������.
		if ((sl[i].iP>-1) && (fabs(sl[i].ap) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB>-1) && (fabs(sl[i].ab) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE>-1) && (fabs(sl[i].ae) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN>-1) && (fabs(sl[i].an) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS>-1) && (fabs(sl[i].as) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT>-1) && (fabs(sl[i].at) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW>-1) && (fabs(sl[i].aw) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB2>-1) && (fabs(sl[i].ab2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE2>-1) && (fabs(sl[i].ae2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN2>-1) && (fabs(sl[i].an2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS2>-1) && (fabs(sl[i].as2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT2>-1) && (fabs(sl[i].at2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW2>-1) && (fabs(sl[i].aw2) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB3>-1) && (fabs(sl[i].ab3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE3>-1) && (fabs(sl[i].ae3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN3>-1) && (fabs(sl[i].an3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS3>-1) && (fabs(sl[i].as3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT3>-1) && (fabs(sl[i].at3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW3>-1) && (fabs(sl[i].aw3) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB4>-1) && (fabs(sl[i].ab4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE4>-1) && (fabs(sl[i].ae4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN4>-1) && (fabs(sl[i].an4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS4>-1) && (fabs(sl[i].as4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT4>-1) && (fabs(sl[i].at4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW4>-1) && (fabs(sl[i].aw4) > nonzeroEPS)) (nna)++;

	}
	for (integer i = 0; i<maxbound; i++) {
		// ��������� ����.
		if ((slb[i].iW>-1) && (fabs(slb[i].aw) > nonzeroEPS)) (nna)++;
		if ((slb[i].iI>-1) && (fabs(slb[i].ai) > nonzeroEPS)) (nna)++;
	}

	integer nnu = 0; // ����� �����������.
	nnu = maxelm + maxbound;

	typedef doublereal    ScalarType;  // feel free to change this to double if supported by your device
	//typedef float    ScalarType;
	typedef cusp::device_memory MemorySpace;


	// allocate storage for (nnu,nnu) matrix with nna nonzeros
	cusp::coo_matrix<int, ScalarType, cusp::host_memory> Ah(nnu, nnu, nna);
	//cusp::coo_matrix<int, float, cusp::device_memory> Ah(nnu, nnu, nna);

	//printf("0\n");
	//getchar();

	// initialize matrix entries on host
	nna = 0;
	//Ah.row_indices[0] = 0; Ah.column_indices[0] = 0; Ah.values[0] = 10.0; // demo interface
	for (integer i = 0; i<maxelm; i++) {
		// ������������ �������.
		if ((sl[i].iP>-1) && (fabs(sl[i].ap) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iP; Ah.values[nna] = sl[i].ap / alpharelax;
			(nna)++;
		}
		if ((sl[i].iB > -1) && (fabs(sl[i].ab) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB; Ah.values[nna] = -sl[i].ab;
			(nna)++;
		}
		if ((sl[i].iE > -1) && (fabs(sl[i].ae) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE; Ah.values[nna] = -sl[i].ae;
			(nna)++;
		}
		if ((sl[i].iN > -1) && (fabs(sl[i].an) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN; Ah.values[nna] = -sl[i].an;
			(nna)++;
		}
		if ((sl[i].iS > -1) && (fabs(sl[i].as) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS; Ah.values[nna] = -sl[i].as;
			(nna)++;
		}
		if ((sl[i].iT > -1) && (fabs(sl[i].at) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT; Ah.values[nna] = -sl[i].at;
			(nna)++;
		}
		if ((sl[i].iW > -1) && (fabs(sl[i].aw) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW; Ah.values[nna] = -sl[i].aw;
			(nna)++;
		}

		if ((sl[i].iB2 > -1) && (fabs(sl[i].ab2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB2; Ah.values[nna] = -sl[i].ab2;
			(nna)++;
		}
		if ((sl[i].iE2 > -1) && (fabs(sl[i].ae2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE2; Ah.values[nna] = -sl[i].ae2;
			(nna)++;
		}
		if ((sl[i].iN2 > -1) && (fabs(sl[i].an2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN2; Ah.values[nna] = -sl[i].an2;
			(nna)++;
		}
		if ((sl[i].iS2 > -1) && (fabs(sl[i].as2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS2; Ah.values[nna] = -sl[i].as2;
			(nna)++;
		}
		if ((sl[i].iT2 > -1) && (fabs(sl[i].at2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT2; Ah.values[nna] = -sl[i].at2;
			(nna)++;
		}
		if ((sl[i].iW2 > -1) && (fabs(sl[i].aw2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW2; Ah.values[nna] = -sl[i].aw2;
			(nna)++;
		}

		if ((sl[i].iB3 > -1) && (fabs(sl[i].ab3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB3; Ah.values[nna] = -sl[i].ab3;
			(nna)++;
		}
		if ((sl[i].iE3 > -1) && (fabs(sl[i].ae3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE3; Ah.values[nna] = -sl[i].ae3;
			(nna)++;
		}
		if ((sl[i].iN3 > -1) && (fabs(sl[i].an3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN3; Ah.values[nna] = -sl[i].an3;
			(nna)++;
		}
		if ((sl[i].iS3 > -1) && (fabs(sl[i].as3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS3; Ah.values[nna] = -sl[i].as3;
			(nna)++;
		}
		if ((sl[i].iT3 > -1) && (fabs(sl[i].at3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT3; Ah.values[nna] = -sl[i].at3;
			(nna)++;
		}
		if ((sl[i].iW3 > -1) && (fabs(sl[i].aw3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW3; Ah.values[nna] = -sl[i].aw3;
			(nna)++;
		}

		if ((sl[i].iB4 > -1) && (fabs(sl[i].ab4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB4; Ah.values[nna] = -sl[i].ab4;
			(nna)++;
		}
		if ((sl[i].iE4 > -1) && (fabs(sl[i].ae4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE4; Ah.values[nna] = -sl[i].ae4;
			(nna)++;
		}
		if ((sl[i].iN4 > -1) && (fabs(sl[i].an4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN4; Ah.values[nna] = -sl[i].an4;
			(nna)++;
		}
		if ((sl[i].iS4 > -1) && (fabs(sl[i].as4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS4; Ah.values[nna] = -sl[i].as4;
			(nna)++;
		}
		if ((sl[i].iT4 > -1) && (fabs(sl[i].at4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT4; Ah.values[nna] = -sl[i].at4;
			(nna)++;
		}
		if ((sl[i].iW4 > -1) && (fabs(sl[i].aw4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW4; Ah.values[nna] = -sl[i].aw4;
			(nna)++;
		}

	}

	for (integer i = 0; i<maxbound; i++) {
		// ��������� ����.
		if ((slb[i].iW>-1) && (fabs(slb[i].aw) > nonzeroEPS)) {
			Ah.row_indices[nna] = slb[i].iW; Ah.column_indices[nna] = slb[i].iW; Ah.values[nna] = slb[i].aw;
			(nna)++;
		}
		if ((slb[i].iI > -1) && (fabs(slb[i].ai) > nonzeroEPS)) {
			Ah.row_indices[nna] = slb[i].iW; Ah.column_indices[nna] = slb[i].iI; Ah.values[nna] = -slb[i].ai;
			(nna)++;
		}
	}


	cusp::array1d<ScalarType, cusp::host_memory> xh(nnu);
	cusp::array1d<ScalarType, cusp::host_memory> yh(nnu);
	//cusp::array1d<float, cusp::device_memory> xh(nnu);
	//cusp::array1d<float, cusp::device_memory> yh(nnu);

	//printf("0i5\n");
	//getchar();

	// yh = Ah x xh;
	for (integer i = 0; i < maxelm + maxbound; i++) {
		// b == dV[i];
		yh[i] = dV[i];
		xh[i] = 0.0;
		//xh[i] = dX0[i];
	}

	//printf("1\n");
	//getchar();
	// convert host->device
	//cusp::array1d<float, cusp::device_memory> xd = xh;
	//cusp::array1d<float, cusp::device_memory> xd(xh);
	//printf("2\n");
	//getchar();
	//cusp::array1d<float, cusp::device_memory> yd = yh;
	//cusp::array1d<float, cusp::device_memory> yd(yh);
	//printf("3\n");
	//getchar();

	//cusp::coo_matrix<int, float, cusp::device_memory> Ad = Ah;
	//cusp::coo_matrix<int, float, cusp::device_memory> Ad(Ah);
	//printf("4\n");
	//getchar();

	// ���������� ��������� 25 ������� 2016.
	doublereal tolerance = 1e-8;
	if (bSIMPLErun_now_for_temperature) {
		// ��� �������� ������� ��� CFD ����� ���� 
		// ������� ���������� �� ������ ��������� �������
		// ��������� ������ (���������� ������� cfd + temperature 
		// + ����������� ��������-����������.).
		switch (iVar) {
		case VX: tolerance = 1e-5;  break; //5e-5
		case VY: tolerance = 1e-5;  break; // 5e-5
		case VZ: tolerance = 1e-5;  break; // 5e-5
		case TEMP: tolerance = 1e-8;  break; // 1e-7
		case PAM: tolerance = 1e-7; break; // 1e-6
		}
	}

	integer imaxiter = 6000;
	if (((adiabatic_vs_heat_transfer_coeff > 0) || (breakRUMBAcalc_for_nonlinear_boundary_condition))) {
		// ��� ���������� �����. ������.
		// 348878024515312.94
		// 1947852997768.51
		imaxiter = 1;
		tolerance = 0.1;
	}

	if (bonly_solid_calculation) {
		tolerance = 1e-6;
		imaxiter = 6000;
	}

	/*
	if ((bonly_solid_calculation)&&(breakRUMBAcalc_for_nonlinear_boundary_condition)) {
	// 1 V - cycle,
	// ������������� �������� ������� 0.1.
	imaxiter = 1;
	tolerance = 0.1;
	getchar();
	}
	*/

	if (bstart7 && (((adiabatic_vs_heat_transfer_coeff > 0) || (breakRUMBAcalc_for_nonlinear_boundary_condition))))
	{
		imaxiter = 1;
		tolerance = 0.1;
		//cusp::relaxation::gauss_seidel(Ah, xh, yh, monitor); // 15_01_2017

		cusp::array1d<ScalarType, cusp::host_memory> rh(nnu);
		cusp::csr_matrix<int, ScalarType, cusp::host_memory> Ah7(Ah);

		// Construct gauss_seidel relaxation class
		cusp::relaxation::gauss_seidel<ScalarType, cusp::host_memory> Mh7(Ah7);
		// Compute initial residual
		cusp::multiply(Ah7, xh, rh);
		cusp::blas::axpy(yh, rh, ScalarType(-1));
		// Construct monitor with stopping criteria of 100 iterations or 1e-4 residual error
		cusp::monitor<ScalarType> monitor(yh, 100, 1e-4, 0, true);
		// Iteratively solve system
		while (!monitor.finished(rh))
		{
			Mh7(Ah7, yh, xh);
			cusp::multiply(Ah7, xh, rh);
			cusp::blas::axpy(yh, rh, ScalarType(-1));
			++monitor;
		}

		//bstart7 = false;

		if (bonly_solid_calculation) {
			// report status
			monitor.print();
		}

		// ����������� ����������.
		for (integer i = 0; i < maxelm + maxbound; i++) {
			//dX0[i] = xh_ret[i];
			dX0[i] = xh[i];
		}

	}
	else {

		// set stopping criteria
		// iteration_limit = 100
		// relative_tolerance = 1e-6
		//  absolute_tolerance = 0
		//  verbose            = true
		//cusp::monitor<float> monitor(yd, 2000, 1e-6, 0, true);

		//cusp::default_monitor<float> monitor(yd, 6000, 1e-12);
		//-->cusp::default_monitor<ScalarType> monitor(yh, 6000, 1e-8);
		cusp::default_monitor<ScalarType> monitor(yh, imaxiter, tolerance);
		//cusp::default_monitor<float> monitor(yd, 2000, 1e-6);


		cusp::array1d<ScalarType, cusp::device_memory> xd(xh);
		cusp::array1d<ScalarType, cusp::device_memory> yd(yh);
		cusp::coo_matrix<int, ScalarType, cusp::device_memory> Ad(Ah);

		// setup preconditioner
		cusp::precond::aggregation::smoothed_aggregation<int, ScalarType, cusp::device_memory> Md(Ad);
		// setup preconditioner
		//host
		//cusp::precond::aggregation::smoothed_aggregation<int, ScalarType, cusp::host_memory> Mh(Ah);
		//end host
		//cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::device_memory> Md(Mh);
		// ������������ �������������������.
		//cusp::precond::diagonal<float, cusp::device_memory> Md(Ad);
		// AINV (NS Bridson).
		//cusp::precond::scaled_bridson_ainv<float, cusp::device_memory> Md(Ad);

		if (bonly_solid_calculation) {
			//Mh.print();
		}

		// solve A * x = y to default tolerance with BiCGStab
		// with preconditioned BiCGStab
		//--->//cusp::krylov::bicgstab(Ad, xd, yd, monitor, Md);
		//cusp::krylov::bicgstab(Ad, xd, yd, monitor);
		//host
		//cusp::krylov::bicgstab(Ah, xh, yh, monitor, Mh); // 15_01_2017
		//end host

		cusp::krylov::bicgstab(Ad, xd, yd, monitor, Md); // 21_10_2017

		cusp::array1d<ScalarType, cusp::host_memory> xh_ret(xd);

		//integer imy_restart_gmres = 20; // 20 � 50, 2000, 4000 �� ��������.
		//cusp::krylov::gmres(Ah, xh, yh, imy_restart_gmres, monitor, Mh); // �� ��������.

		if (bonly_solid_calculation) {
			// report status
			monitor.print();
		}

		// ����������� ����������.
		for (integer i = 0; i < maxelm + maxbound; i++) {
			dX0[i] = xh_ret[i];
			//dX0[i] = xh[i];
		}

	}

	//cusp::array1d<float, cusp::host_memory> xh_ret = xd;
	//cusp::array1d<float, cusp::host_memory> xh_ret(xd);
	//printf("5\n");
	//getchar();

} // cusp_solver_amgdevice

// ��� ����� ������������� �������� ������ �������� �������������� ���������.
// ���������� Cusp ������ 0.5.1. ��� ���������� � �������� �������� ����� ������������������ 
// �� open Source �������� Apache license 2.0. 
// �� ���������� (����������� ����������) ��������� ����� bicgstab � AINV (NS Brigson) �����
// � �������� �������������������.
// ���� ������������� 12 ������� 2016 ����.
void cusp_solver(equation3D* &sl, equation3D_bon* &slb,
	integer maxelm, integer maxbound,
	doublereal *dV, doublereal* &dX0, integer maxit,
	doublereal alpharelax, integer iVar)
{

	// maxit,  iVar - �� ������������.


	if (dX0 == NULL) {
		dX0 = new doublereal[maxelm + maxbound];
		for (integer i = 0; i < maxelm + maxbound; i++) {
			dX0[i] = 0.0;
		}
	}

	// TODO �������� val, col_ind, row_ptr
	integer nna = 0; // ���������� ��������� ��������� � ������� ����.

	const doublereal nonzeroEPS = 1e-37; // ��� ��������� ������������� ����

	// ������� ����� ��������� ��������� � �������.
	nna = 0;
	for (integer i = 0; i<maxelm; i++) {
		// ������������ �������.
		if ((sl[i].iP>-1) && (fabs(sl[i].ap) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB>-1) && (fabs(sl[i].ab) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE>-1) && (fabs(sl[i].ae) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN>-1) && (fabs(sl[i].an) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS>-1) && (fabs(sl[i].as) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT>-1) && (fabs(sl[i].at) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW>-1) && (fabs(sl[i].aw) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB2>-1) && (fabs(sl[i].ab2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE2>-1) && (fabs(sl[i].ae2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN2>-1) && (fabs(sl[i].an2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS2>-1) && (fabs(sl[i].as2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT2>-1) && (fabs(sl[i].at2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW2>-1) && (fabs(sl[i].aw2) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB3>-1) && (fabs(sl[i].ab3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE3>-1) && (fabs(sl[i].ae3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN3>-1) && (fabs(sl[i].an3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS3>-1) && (fabs(sl[i].as3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT3>-1) && (fabs(sl[i].at3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW3>-1) && (fabs(sl[i].aw3) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB4>-1) && (fabs(sl[i].ab4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE4>-1) && (fabs(sl[i].ae4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN4>-1) && (fabs(sl[i].an4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS4>-1) && (fabs(sl[i].as4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT4>-1) && (fabs(sl[i].at4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW4>-1) && (fabs(sl[i].aw4) > nonzeroEPS)) (nna)++;

	}
	for (integer i = 0; i<maxbound; i++) {
		// ��������� ����.
		if ((slb[i].iW>-1) && (fabs(slb[i].aw) > nonzeroEPS)) (nna)++;
		if ((slb[i].iI>-1) && (fabs(slb[i].ai) > nonzeroEPS)) (nna)++;
	}

	integer nnu = 0; // ����� �����������.
	nnu = maxelm + maxbound;

	typedef doublereal    ScalarType;  // feel free to change this to double if supported by your device
	//typedef float    ScalarType;

	// allocate storage for (nnu,nnu) matrix with nna nonzeros
	cusp::coo_matrix<int, ScalarType, cusp::host_memory> Ah(nnu, nnu, nna);
	//cusp::coo_matrix<int, float, cusp::device_memory> Ah(nnu, nnu, nna);

	//printf("0\n");
	//getchar();

	// initialize matrix entries on host
	nna = 0;
	//Ah.row_indices[0] = 0; Ah.column_indices[0] = 0; Ah.values[0] = 10.0; // demo interface
	for (integer i = 0; i<maxelm; i++) {
		// ������������ �������.
		if ((sl[i].iP>-1) && (fabs(sl[i].ap) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iP; Ah.values[nna] = sl[i].ap / alpharelax;
			(nna)++;
		}
		if ((sl[i].iB > -1) && (fabs(sl[i].ab) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB; Ah.values[nna] = -sl[i].ab;
			(nna)++;
		}
		if ((sl[i].iE > -1) && (fabs(sl[i].ae) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE; Ah.values[nna] = -sl[i].ae;
			(nna)++;
		}
		if ((sl[i].iN > -1) && (fabs(sl[i].an) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN; Ah.values[nna] = -sl[i].an;
			(nna)++;
		}
		if ((sl[i].iS > -1) && (fabs(sl[i].as) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS; Ah.values[nna] = -sl[i].as;
			(nna)++;
		}
		if ((sl[i].iT > -1) && (fabs(sl[i].at) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT; Ah.values[nna] = -sl[i].at;
			(nna)++;
		}
		if ((sl[i].iW > -1) && (fabs(sl[i].aw) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW; Ah.values[nna] = -sl[i].aw;
			(nna)++;
		}

		if ((sl[i].iB2 > -1) && (fabs(sl[i].ab2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB2; Ah.values[nna] = -sl[i].ab2;
			(nna)++;
		}
		if ((sl[i].iE2 > -1) && (fabs(sl[i].ae2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE2; Ah.values[nna] = -sl[i].ae2;
			(nna)++;
		}
		if ((sl[i].iN2 > -1) && (fabs(sl[i].an2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN2; Ah.values[nna] = -sl[i].an2;
			(nna)++;
		}
		if ((sl[i].iS2 > -1) && (fabs(sl[i].as2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS2; Ah.values[nna] = -sl[i].as2;
			(nna)++;
		}
		if ((sl[i].iT2 > -1) && (fabs(sl[i].at2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT2; Ah.values[nna] = -sl[i].at2;
			(nna)++;
		}
		if ((sl[i].iW2 > -1) && (fabs(sl[i].aw2) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW2; Ah.values[nna] = -sl[i].aw2;
			(nna)++;
		}

		if ((sl[i].iB3 > -1) && (fabs(sl[i].ab3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB3; Ah.values[nna] = -sl[i].ab3;
			(nna)++;
		}
		if ((sl[i].iE3 > -1) && (fabs(sl[i].ae3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE3; Ah.values[nna] = -sl[i].ae3;
			(nna)++;
		}
		if ((sl[i].iN3 > -1) && (fabs(sl[i].an3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN3; Ah.values[nna] = -sl[i].an3;
			(nna)++;
		}
		if ((sl[i].iS3 > -1) && (fabs(sl[i].as3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS3; Ah.values[nna] = -sl[i].as3;
			(nna)++;
		}
		if ((sl[i].iT3 > -1) && (fabs(sl[i].at3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT3; Ah.values[nna] = -sl[i].at3;
			(nna)++;
		}
		if ((sl[i].iW3 > -1) && (fabs(sl[i].aw3) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW3; Ah.values[nna] = -sl[i].aw3;
			(nna)++;
		}

		if ((sl[i].iB4 > -1) && (fabs(sl[i].ab4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iB4; Ah.values[nna] = -sl[i].ab4;
			(nna)++;
		}
		if ((sl[i].iE4 > -1) && (fabs(sl[i].ae4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iE4; Ah.values[nna] = -sl[i].ae4;
			(nna)++;
		}
		if ((sl[i].iN4 > -1) && (fabs(sl[i].an4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iN4; Ah.values[nna] = -sl[i].an4;
			(nna)++;
		}
		if ((sl[i].iS4 > -1) && (fabs(sl[i].as4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iS4; Ah.values[nna] = -sl[i].as4;
			(nna)++;
		}
		if ((sl[i].iT4 > -1) && (fabs(sl[i].at4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iT4; Ah.values[nna] = -sl[i].at4;
			(nna)++;
		}
		if ((sl[i].iW4 > -1) && (fabs(sl[i].aw4) > nonzeroEPS)) {
			Ah.row_indices[nna] = sl[i].iP; Ah.column_indices[nna] = sl[i].iW4; Ah.values[nna] = -sl[i].aw4;
			(nna)++;
		}

	}

	for (integer i = 0; i<maxbound; i++) {
		// ��������� ����.
		if ((slb[i].iW>-1) && (fabs(slb[i].aw) > nonzeroEPS)) {
			Ah.row_indices[nna] = slb[i].iW; Ah.column_indices[nna] = slb[i].iW; Ah.values[nna] = slb[i].aw;
			(nna)++;
		}
		if ((slb[i].iI > -1) && (fabs(slb[i].ai) > nonzeroEPS)) {
			Ah.row_indices[nna] = slb[i].iW; Ah.column_indices[nna] = slb[i].iI; Ah.values[nna] = -slb[i].ai;
			(nna)++;
		}
	}


	cusp::array1d<ScalarType, cusp::host_memory> xh(nnu);
	cusp::array1d<ScalarType, cusp::host_memory> yh(nnu);
	//cusp::array1d<float, cusp::device_memory> xh(nnu);
	//cusp::array1d<float, cusp::device_memory> yh(nnu);

	//printf("0i5\n");
	//getchar();

	// yh = Ah x xh;
	for (integer i = 0; i < maxelm + maxbound; i++) {
		// b == dV[i];
		yh[i] = dV[i];
		xh[i] = 0.0;
		//xh[i] = dX0[i];
	}

	//printf("1\n");
	//getchar();
	// convert host->device
	//cusp::array1d<float, cusp::device_memory> xd = xh;
	cusp::array1d<ScalarType, cusp::device_memory> xd(xh);
	//printf("2\n");
	//getchar();
	//cusp::array1d<float, cusp::device_memory> yd = yh;
	cusp::array1d<ScalarType, cusp::device_memory> yd(yh);
	//printf("3\n");
	//getchar();

	//cusp::coo_matrix<int, float, cusp::device_memory> Ad = Ah;
	cusp::coo_matrix<int, ScalarType, cusp::device_memory> Ad(Ah);
	//printf("4\n");
	//getchar();


	// set stopping criteria
	// iteration_limit = 100
	// relative_tolerance = 1e-6
	//  absolute_tolerance = 0
	//  verbose            = true
	//cusp::monitor<float> monitor(yd, 2000, 1e-6, 0, true);

	//cusp::default_monitor<float> monitor(yd, 6000, 1e-12);
	// 1e-6 ���������� ��� ������������ �����.
	cusp::default_monitor<ScalarType> monitor(yh, 6000, 1e-8);
	//cusp::default_monitor<float> monitor(yd, 2000, 1e-6);

	// setup preconditioner
	//cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::device_memory> Md(Ad);
	// setup preconditioner
	//cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::host_memory> Mh(Ah);
	//cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::device_memory> Md(Mh);
	// ������������ �������������������.
	//cusp::precond::diagonal<float, cusp::device_memory> Md(Ad);
	// AINV (NS Bridson).
	cusp::precond::scaled_bridson_ainv<ScalarType, cusp::device_memory> Md(Ad);


	// solve A * x = y to default tolerance with BiCGStab
	// with preconditioned BiCGStab
	cusp::krylov::bicgstab(Ad, xd, yd, monitor, Md);
	//cusp::krylov::bicgstab(Ad, xd, yd, monitor);
	//cusp::krylov::bicgstab(Ah, xh, yh, monitor, Mh);



	//cusp::array1d<float, cusp::host_memory> xh_ret = xd;
	cusp::array1d<ScalarType, cusp::host_memory> xh_ret(xd);
	//printf("5\n");
	//getchar();

	// ����������� ����������.
	for (integer i = 0; i < maxelm + maxbound; i++) {
		dX0[i] = xh_ret[i];
		//dX0[i] = xh[i];
	}

} // cusp_solver

bool bcusp_gl_first_start = true;
cusp::coo_matrix<int, doublereal, cusp::host_memory> Ah_gl;
cusp::array1d<doublereal, cusp::host_memory> xh_gl;
cusp::array1d<doublereal, cusp::host_memory> yh_gl;
cusp::array1d<doublereal, cusp::device_memory> xd_gl;
cusp::array1d<doublereal, cusp::device_memory> yd_gl;
cusp::coo_matrix<int, doublereal, cusp::device_memory> Ad_gl;

// ��� ����� ������������� �������� ������ �������� �������������� ���������.
// ���������� Cusp ������ 0.5.1. ��� ���������� � �������� �������� ����� ������������������ 
// �� open Source �������� Apache license 2.0. 
// �� ���������� (����������� ����������) ��������� ����� bicgstab � AINV (NS Brigson) �����
// � �������� �������������������.
// ���� ������������� 12 ������� 2016 ����.
void cusp_solver_global_allocate(equation3D* &sl, equation3D_bon* &slb,
	integer maxelm, integer maxbound,
	doublereal *dV, doublereal* &dX0, integer maxit,
	doublereal alpharelax, integer iVar)
{

	// maxit,  iVar - �� ������������.


	if (dX0 == NULL) {
		dX0 = new doublereal[maxelm + maxbound];
		for (integer i = 0; i < maxelm + maxbound; i++) {
			dX0[i] = 0.0;
		}
	}

	// TODO �������� val, col_ind, row_ptr
	integer nna = 0; // ���������� ��������� ��������� � ������� ����.

	const doublereal nonzeroEPS = 1e-37; // ��� ��������� ������������� ����

	// ������� ����� ��������� ��������� � �������.
	nna = 0;
	for (integer i = 0; i<maxelm; i++) {
		// ������������ �������.
		if ((sl[i].iP>-1) && (fabs(sl[i].ap) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB>-1) && (fabs(sl[i].ab) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE>-1) && (fabs(sl[i].ae) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN>-1) && (fabs(sl[i].an) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS>-1) && (fabs(sl[i].as) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT>-1) && (fabs(sl[i].at) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW>-1) && (fabs(sl[i].aw) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB2>-1) && (fabs(sl[i].ab2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE2>-1) && (fabs(sl[i].ae2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN2>-1) && (fabs(sl[i].an2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS2>-1) && (fabs(sl[i].as2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT2>-1) && (fabs(sl[i].at2) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW2>-1) && (fabs(sl[i].aw2) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB3>-1) && (fabs(sl[i].ab3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE3>-1) && (fabs(sl[i].ae3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN3>-1) && (fabs(sl[i].an3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS3>-1) && (fabs(sl[i].as3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT3>-1) && (fabs(sl[i].at3) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW3>-1) && (fabs(sl[i].aw3) > nonzeroEPS)) (nna)++;

		if ((sl[i].iB4>-1) && (fabs(sl[i].ab4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iE4>-1) && (fabs(sl[i].ae4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iN4>-1) && (fabs(sl[i].an4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iS4>-1) && (fabs(sl[i].as4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iT4>-1) && (fabs(sl[i].at4) > nonzeroEPS)) (nna)++;
		if ((sl[i].iW4>-1) && (fabs(sl[i].aw4) > nonzeroEPS)) (nna)++;

	}
	for (integer i = 0; i<maxbound; i++) {
		// ��������� ����.
		if ((slb[i].iW>-1) && (fabs(slb[i].aw) > nonzeroEPS)) (nna)++;
		if ((slb[i].iI>-1) && (fabs(slb[i].ai) > nonzeroEPS)) (nna)++;
	}

	integer nnu = 0; // ����� �����������.
	nnu = maxelm + maxbound;

	typedef doublereal    ScalarType;  // feel free to change this to double if supported by your device
	//typedef float    ScalarType;

	// allocate storage for (nnu,nnu) matrix with nna nonzeros

	/*
	cusp::coo_matrix<int, ScalarType, cusp::host_memory> Ah(nnu, nnu, nna);

	*/
	if (bcusp_gl_first_start) {
		Ah_gl.resize(nnu, nnu, nna);
	}
	//cusp::coo_matrix<int, float, cusp::device_memory> Ah(nnu, nnu, nna);

	//printf("0\n");
	//getchar();

	// initialize matrix entries on host
	nna = 0;
	//Ah_gl.row_indices[0] = 0; Ah_gl.column_indices[0] = 0; Ah_gl.values[0] = 10.0; // demo interface
	for (integer i = 0; i<maxelm; i++) {
		// ������������ �������.
		if ((sl[i].iP>-1) && (fabs(sl[i].ap) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iP; Ah_gl.values[nna] = sl[i].ap / alpharelax;
			(nna)++;
		}
		if ((sl[i].iB > -1) && (fabs(sl[i].ab) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iB; Ah_gl.values[nna] = -sl[i].ab;
			(nna)++;
		}
		if ((sl[i].iE > -1) && (fabs(sl[i].ae) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iE; Ah_gl.values[nna] = -sl[i].ae;
			(nna)++;
		}
		if ((sl[i].iN > -1) && (fabs(sl[i].an) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iN; Ah_gl.values[nna] = -sl[i].an;
			(nna)++;
		}
		if ((sl[i].iS > -1) && (fabs(sl[i].as) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iS; Ah_gl.values[nna] = -sl[i].as;
			(nna)++;
		}
		if ((sl[i].iT > -1) && (fabs(sl[i].at) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iT; Ah_gl.values[nna] = -sl[i].at;
			(nna)++;
		}
		if ((sl[i].iW > -1) && (fabs(sl[i].aw) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iW; Ah_gl.values[nna] = -sl[i].aw;
			(nna)++;
		}

		if ((sl[i].iB2 > -1) && (fabs(sl[i].ab2) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iB2; Ah_gl.values[nna] = -sl[i].ab2;
			(nna)++;
		}
		if ((sl[i].iE2 > -1) && (fabs(sl[i].ae2) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iE2; Ah_gl.values[nna] = -sl[i].ae2;
			(nna)++;
		}
		if ((sl[i].iN2 > -1) && (fabs(sl[i].an2) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iN2; Ah_gl.values[nna] = -sl[i].an2;
			(nna)++;
		}
		if ((sl[i].iS2 > -1) && (fabs(sl[i].as2) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iS2; Ah_gl.values[nna] = -sl[i].as2;
			(nna)++;
		}
		if ((sl[i].iT2 > -1) && (fabs(sl[i].at2) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iT2; Ah_gl.values[nna] = -sl[i].at2;
			(nna)++;
		}
		if ((sl[i].iW2 > -1) && (fabs(sl[i].aw2) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iW2; Ah_gl.values[nna] = -sl[i].aw2;
			(nna)++;
		}

		if ((sl[i].iB3 > -1) && (fabs(sl[i].ab3) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iB3; Ah_gl.values[nna] = -sl[i].ab3;
			(nna)++;
		}
		if ((sl[i].iE3 > -1) && (fabs(sl[i].ae3) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iE3; Ah_gl.values[nna] = -sl[i].ae3;
			(nna)++;
		}
		if ((sl[i].iN3 > -1) && (fabs(sl[i].an3) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iN3; Ah_gl.values[nna] = -sl[i].an3;
			(nna)++;
		}
		if ((sl[i].iS3 > -1) && (fabs(sl[i].as3) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iS3; Ah_gl.values[nna] = -sl[i].as3;
			(nna)++;
		}
		if ((sl[i].iT3 > -1) && (fabs(sl[i].at3) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iT3; Ah_gl.values[nna] = -sl[i].at3;
			(nna)++;
		}
		if ((sl[i].iW3 > -1) && (fabs(sl[i].aw3) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iW3; Ah_gl.values[nna] = -sl[i].aw3;
			(nna)++;
		}

		if ((sl[i].iB4 > -1) && (fabs(sl[i].ab4) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iB4; Ah_gl.values[nna] = -sl[i].ab4;
			(nna)++;
		}
		if ((sl[i].iE4 > -1) && (fabs(sl[i].ae4) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iE4; Ah_gl.values[nna] = -sl[i].ae4;
			(nna)++;
		}
		if ((sl[i].iN4 > -1) && (fabs(sl[i].an4) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iN4; Ah_gl.values[nna] = -sl[i].an4;
			(nna)++;
		}
		if ((sl[i].iS4 > -1) && (fabs(sl[i].as4) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iS4; Ah_gl.values[nna] = -sl[i].as4;
			(nna)++;
		}
		if ((sl[i].iT4 > -1) && (fabs(sl[i].at4) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iT4; Ah_gl.values[nna] = -sl[i].at4;
			(nna)++;
		}
		if ((sl[i].iW4 > -1) && (fabs(sl[i].aw4) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = sl[i].iP; Ah_gl.column_indices[nna] = sl[i].iW4; Ah_gl.values[nna] = -sl[i].aw4;
			(nna)++;
		}

	}

	for (integer i = 0; i<maxbound; i++) {
		// ��������� ����.
		if ((slb[i].iW>-1) && (fabs(slb[i].aw) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = slb[i].iW; Ah_gl.column_indices[nna] = slb[i].iW; Ah_gl.values[nna] = slb[i].aw;
			(nna)++;
		}
		if ((slb[i].iI > -1) && (fabs(slb[i].ai) > nonzeroEPS)) {
			Ah_gl.row_indices[nna] = slb[i].iW; Ah_gl.column_indices[nna] = slb[i].iI; Ah_gl.values[nna] = -slb[i].ai;
			(nna)++;
		}
	}

	/*
	cusp::array1d<ScalarType, cusp::host_memory> xh(nnu);
	cusp::array1d<ScalarType, cusp::host_memory> yh(nnu);
	*/
	if (bcusp_gl_first_start) {
		xh_gl.resize(nnu);
		yh_gl.resize(nnu);
	}
	//cusp::array1d<float, cusp::device_memory> xh(nnu);
	//cusp::array1d<float, cusp::device_memory> yh(nnu);

	//printf("0i5\n");
	//getchar();

	// yh = Ah x xh;
	for (integer i = 0; i < maxelm + maxbound; i++) {
		// b == dV[i];
		yh_gl[i] = dV[i];
		xh_gl[i] = 0.0;
		//xh[i] = dX0[i];
	}

	/*
	//printf("1\n");
	//getchar();
	// convert host->device
	//cusp::array1d<float, cusp::device_memory> xd = xh;
	cusp::array1d<ScalarType, cusp::device_memory> xd(xh);
	//printf("2\n");
	//getchar();
	//cusp::array1d<float, cusp::device_memory> yd = yh;
	cusp::array1d<ScalarType, cusp::device_memory> yd(yh);
	//printf("3\n");
	//getchar();

	//cusp::coo_matrix<int, float, cusp::device_memory> Ad = Ah;
	cusp::coo_matrix<int, ScalarType, cusp::device_memory> Ad(Ah);
	//printf("4\n");
	//getchar();
	*/
	if (bcusp_gl_first_start) {
		xd_gl.resize(nnu);
		yd_gl.resize(nnu);
		Ad_gl.resize(nnu, nnu, nna);
	}

	cusp::copy(xh_gl, xd_gl);
	cusp::copy(yh_gl, yd_gl);
	cusp::copy(Ah_gl, Ad_gl);

	// set stopping criteria
	// iteration_limit = 100
	// relative_tolerance = 1e-6
	//  absolute_tolerance = 0
	//  verbose            = true
	//cusp::monitor<float> monitor(yd, 2000, 1e-6, 0, true);

	//cusp::default_monitor<float> monitor(yd, 6000, 1e-12);
	// 1e-6 ���������� ��� ������������ �����.
	cusp::default_monitor<ScalarType> monitor(yh_gl, 6000, 1e-8);
	//cusp::default_monitor<float> monitor(yd, 2000, 1e-6);

	// setup preconditioner
	//cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::device_memory> Md(Ad);
	// setup preconditioner
	//cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::host_memory> Mh(Ah);
	//cusp::precond::aggregation::smoothed_aggregation<int, float, cusp::device_memory> Md(Mh);
	// ������������ �������������������.
	//cusp::precond::diagonal<float, cusp::device_memory> Md(Ad);
	// AINV (NS Bridson).
	cusp::precond::scaled_bridson_ainv<ScalarType, cusp::device_memory> Md(Ad_gl);


	// solve A * x = y to default tolerance with BiCGStab
	// with preconditioned BiCGStab
	cusp::krylov::bicgstab(Ad_gl, xd_gl, yd_gl, monitor, Md);
	//cusp::krylov::bicgstab(Ad, xd, yd, monitor);
	//cusp::krylov::bicgstab(Ah, xh, yh, monitor, Mh);



	//cusp::array1d<float, cusp::host_memory> xh_ret = xd;
	//cusp::array1d<ScalarType, cusp::host_memory> xh_ret(xd);
	cusp::copy(xd_gl, xh_gl);
	//printf("5\n");
	//getchar();

	// ����������� ����������.
	for (integer i = 0; i < maxelm + maxbound; i++) {
		//dX0[i] = xh_ret[i];
		dX0[i] = xh_gl[i];
	}

	bcusp_gl_first_start = false;

} // cusp_solver_global_allocate

#endif